 /****u* CUDA/1Dconv_new.cu
**
** 
** AUTHOR
**  Luigi Genovese
**
** SOURCE
*/
  
#include <stdio.h>
#include "commonDef.h"


//parameters for the kernel (global variables)
//__constant__ int keys[MAX_CONSTANT_SIZE/sizeof(int)];

#include "kernels_compress.hcu"


template<typename T>
int uncompressgpu(int n1, int n2, int n3,
		  T *psicf,T *psig, int *keys)
{


  //decide also the number of threads and block of the grid
  dim3  gridC(nblocksC, 1, 1);  
  dim3  threadsC(ELEMS_BLOCK, nseg_blockC , 1);

  //set the value of the psig array to zero
  hipMemset((void*) psig,0,8*n1*n2*n3*sizeof(T));

  uncompresscoarsefine<T> <<< gridC, threadsC >>>(n1,n2,n3,psicf,psig,keys);
  hipDeviceSynchronize();

  return 0;

}

template<typename T>
int compressgpu(int n1, int n2, int n3, 
		T *psig,T *psicf, int *keys)
{


  //decide also the number of threads and block of the grid
  dim3  gridC(nblocksC, 1, 1);  
  dim3  threadsC(ELEMS_BLOCK, nseg_blockC , 1);

  compresscoarsefine<T> <<< gridC, threadsC >>>(n1,n2,n3,psig,psicf,keys);
  hipDeviceSynchronize();

  return 0;

}




