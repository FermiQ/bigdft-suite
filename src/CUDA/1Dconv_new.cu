#include "hip/hip_runtime.h"
/****u* CUDA/1Dconv_new.cu
**
** 
** AUTHOR
**  Luigi Genovese
**
** SOURCE
*/

#include <stdio.h>
#include <cutil.h>
#include <pthread.h>
#include <semaphore.h>
#include <sched.h>

#define max(a,b) (a > b ? a : b)
#define min(a,b) (a < b ? a : b)

//maximum size of the shared memory array
//conceived for maximize occupancy on a hardware of compute
//capability 1.2 and higher (1024 threads at same time on a given multiprocessor)
#define MAX_SHARED_SIZE 3072 //16*256 4 kB (should be =~ 3.9 kB, try also 3072)
#define HALF_WARP_SIZE 16 // for all architectures
#define NUM_LINES 16 
#define HW_ELEM 1 //this is HALF_WARP_SIZE/NUM_LINES

typedef struct  _par
{
  //unsigned int LinesPerBlock;
  unsigned int ElementsPerBlock;
  //unsigned int ElementsPerHalfWarp;

  int lowfil, lupfil; //structure of f_fct
  int thline[HALF_WARP_SIZE]; //line considered by a thread within the half-warp
  int thelem[HALF_WARP_SIZE]; //elements considered by a thread within the half-warp
  int hwelem_calc[16]; //maximum number of half warps
  int hwelem_copy[16]; //maximum number of half-warps
  int hwoffset_calc[16]; //maximum number of half warps
  int hwoffset_copy[16]; //maximum number of half-warps
  
  float fil[16];

} par_t;

__constant__ par_t par;

//declare the texture for binding the input psi
//texture<float> psi_tex;


int dogenconv(int ndat,
	      int n, 
	      float *GPU_idata,
	      float *GPU_odata,
	      int lowfil,
	      int lupfil);

void correctSequence(int thds,int elem,int * tab);

//create the parameters to be used for calculating the convolution
//with a given stride
void constantParameters(par_t* par,
			unsigned int* num_halfwarps,
			//unsigned int num_lines,
			int n,
			int ndat,
			int lowfil, //lowfil + lupfil must be a multiple of 16
			int lupfil,
			unsigned int* linecuts,
			unsigned int* num_blocks)

{

  //number of lines treated by each block
  //par->LinesPerBlock = num_lines;

  //number of total allowed elements of a input line
  unsigned int num_elem_tot=MAX_SHARED_SIZE/sizeof(float)/NUM_LINES; //between1024and64
  //unsigned int num_elem_tot=MAX_SHARED_SIZE/sizeof(float)/num_lines; //between 1024 and 64
  
  //number of elements of the output
  unsigned int num_elem_max=min(num_elem_tot-lowfil-lupfil-1,n); //between 1008 and 48 for 16-fil

  //number of elements treated by the single half-warp
  //par -> ElementsPerHalfWarp = HALF_WARP_SIZE/num_lines; //it is assumed they are multiples

  //number of pieces in which a line is divided
  //if the line is too small and not a multiple of ElementsPerHalfWarp
  //divide the line in two
  *linecuts=
    //(n <= num_elem_max && n % par -> ElementsPerHalfWarp !=0 ? 2 : (n-1)/num_elem_max+1);
    (n <= num_elem_max && n % HW_ELEM !=0 ? 2 : (n-1)/num_elem_max+1);

  //number of blocks in ndat direction
  //*num_blocks=((ndat-1)/num_lines + 1);
  *num_blocks=((ndat-1)/NUM_LINES + 1);

  //printf("num_elem_tot %i,num_elem_max %i,linecuts %i,num_blocks %i,elemperHW %i \n",
  //num_elem_tot,num_elem_max,*linecuts,*num_blocks, par -> ElementsPerHalfWarp);

  //number of elements treated by each block 
  //this may pose problems for values of n dimensions less than 48
  //when n is not a multiple of ElementsPerHalfWarp
  par->ElementsPerBlock = 
    //min(par->ElementsPerHalfWarp*(((n-1)/(*linecuts))/par->ElementsPerHalfWarp+1),n);
    min(HW_ELEM*(((n-1)/(*linecuts))/HW_ELEM+1),n);

  int halfwarps=16;
  //calculate the maximum number of halfwarps (between 4 and 16)
  for(int i =3; i>=0; --i)
    {
      //if(par->ElementsPerBlock/par->ElementsPerHalfWarp >= 1 << i)
      if(par->ElementsPerBlock/HW_ELEM >= 1 << i)
	{
	  halfwarps = 1 << i;
	  break;
	}
    }

  *num_halfwarps = halfwarps;

  for(int j=0;j < HALF_WARP_SIZE ; ++j)
    {
      //par->thline[j]= j & (num_lines - 1); //num_lines always a power of two 
      //par->thelem[j]= j / num_lines; 

      par->thline[j]= j & (NUM_LINES - 1); //num_lines always a power of two 
      par->thelem[j]= j / NUM_LINES; 
    }

  //define the sequences of the number of elements
  correctSequence(halfwarps,par->ElementsPerBlock/HW_ELEM,par->hwelem_calc);

  correctSequence(halfwarps,(par->ElementsPerBlock+lowfil+lupfil+1)/HW_ELEM,
		  par->hwelem_copy);

  //correctSequence(halfwarps,par->ElementsPerBlock/par->ElementsPerHalfWarp,
  //par->hwelem_calc);

  //correctSequence(halfwarps,(par->ElementsPerBlock+lowfil+lupfil+1)/par->ElementsPerHalfWarp,
  //par->hwelem_copy);


  //define the offsets
  for(int j=0,pos_calc=0,pos_copy=0;j < halfwarps ; ++j)
    {
      par->hwoffset_calc[j]=pos_calc;
      par->hwoffset_copy[j]=pos_copy;
      pos_calc+=HW_ELEM*par->hwelem_calc[j];
      pos_copy+=HW_ELEM*par->hwelem_copy[j];
      //pos_calc+=par->ElementsPerHalfWarp*par->hwelem_calc[j];
      //pos_copy+=par->ElementsPerHalfWarp*par->hwelem_copy[j];

    }
 
  //lowfil and lupfil parameters
  par->lowfil = lowfil;
  par->lupfil = lupfil;

  //printf("ElementsPerBlock %i,HalfWarpCalculatedElements %i,HalfWarpCopiedElements %i,LastHalfWarpCalcElements %i, LastHalfWarpCopiedElements %i \n",
  //par->ElementsPerBlock,par->hwelem_calc[0],par->hwelem_copy[0],
  //par->hwelem_calc[halfwarps-1],par->hwelem_copy[halfwarps-1]);

  //filter values for this convolution, hard coded
  par->fil[0] = 8.4334247333529341094733325815816e-7f;
  par->fil[1] =-0.1290557201342060969516786758559028e-4f;
  par->fil[2] = 0.8762984476210559564689161894116397e-4f;
  par->fil[3] =-0.30158038132690463167163703826169879e-3f;
  par->fil[4] = 0.174723713672993903449447812749852942e-2f;
  par->fil[5] =-0.942047030201080385922711540948195075e-2f;
  par->fil[6] = 0.2373821463724942397566389712597274535e-1f;
  par->fil[7] = 0.612625895831207982195380597e-1f;
  par->fil[8] = 0.9940415697834003993178616713f;
  par->fil[9] =-0.604895289196983516002834636e-1f;
  par->fil[10]=-0.2103025160930381434955489412839065067e-1f;
  par->fil[11]= 0.1337263414854794752733423467013220997e-1f;
  par->fil[12]=-0.344128144493493857280881509686821861e-2f;
  par->fil[13]= 0.49443227688689919192282259476750972e-3f;
  par->fil[14]=-0.5185986881173432922848639136911487e-4f;
  par->fil[15]= 2.72734492911979659657715313017228e-6f;


}

//uniformise the tabular of the number of elements treated by each
//thread (counterpart of uniformiseTab)
void correctSequence(int thds,int elem,int * tab)
{
  //put to zero all the values;
  for(int j=0;j< elem; ++j)
    {
      tab[j]=0;
    }

  //then start to fill consecutively until reaching of the end
  //if elem > thds no element will be zero
  //this is the most balanced choice
  for(int i=0;i< elem; ++i)
    {
      tab[i % thds]+=1;
    }
}



//1D convolution of multiple lines in the same block
//__global__ void conv1d_stride(int n,int ndat, float *psi_out)
__global__ void conv1d_stride(int n,int ndat, float *psi_in, float *psi_out)
{

  //line treated by the given block
  unsigned int lineOffset = min(blockIdx.y*NUM_LINES,ndat-NUM_LINES);
  //starting element treated by the block
  unsigned int elemOffset = min(blockIdx.x*par.ElementsPerBlock,n-par.ElementsPerBlock);

  //line treated by the given block
  //unsigned int lineOffset = min(blockIdx.y*par.LinesPerBlock,ndat-par.LinesPerBlock);
  //starting element treated by the block
  //unsigned int elemOffset = min(blockIdx.x*par.ElementsPerBlock,n-par.ElementsPerBlock);

  //half-warp id
  const unsigned int hwid = threadIdx.y;
  //tid within the HW
  const unsigned int tid_hw = threadIdx.x;

  //shared memory array
  __shared__ float psi_sh[MAX_SHARED_SIZE/sizeof(float)];

  //line treated by the given thread in ndat axis
  //which is the input base element
  unsigned int BaseElem = par.thline[tid_hw] + lineOffset;
  //write data in shared memory
  //element treated by the given thread in n-axis
  unsigned int thelem = par.thelem[tid_hw] + par.hwoffset_copy[hwid];

  unsigned int ShBaseElem = tid_hw + NUM_LINES*par.hwoffset_copy[hwid];
  //unsigned int ShBaseElem = tid_hw + par.LinesPerBlock*par.hwoffset_copy[hwid];

  int epsilon,npos;

  //NOTE: it is assumed that for non-first segments the starting
  //points is far enough for the filter to be contained
  //and the same for non-last segments.
  //in other terms: lenght of the line is always bigger than
  //max(lowfil,lupfil)

  for(int i=0,ipos=elemOffset-par.lowfil+thelem;i < par.hwelem_copy[hwid] ; ++i)
    {
      epsilon=(ipos < 0 ? -1 : ipos/n);
      npos=ipos-epsilon*n;
      psi_sh[ShBaseElem]=psi_in[BaseElem+ndat*npos];
      //psi_sh[ShBaseElem]=tex1Dfetch(psi_tex,BaseElem+ndat*npos);

      ShBaseElem += HALF_WARP_SIZE;
      ipos += HW_ELEM;
      //ipos += par.ElementsPerHalfWarp;
      
    }

  //end shared memory copy
  __syncthreads();

  //element treated by the given thread in n-axis
  thelem = par.thelem[tid_hw] + par.hwoffset_calc[hwid];
  //base element for the given thread in shared memory
  ShBaseElem = tid_hw + NUM_LINES*par.hwoffset_calc[hwid];
  //ShBaseElem = tid_hw + par.LinesPerBlock*par.hwoffset_calc[hwid];

  //output base element, from the input one
  BaseElem =  n*BaseElem+ thelem + elemOffset;

  //perform convolution in shared memory 
  //each thread calculate a number of elements, identical for each
  //half-warp
  //#pragma unroll 5 (to be tested if it is important)

  for(int i=0;i < par.hwelem_calc[hwid]; ++i)
    {
      //values of the convolution
      register float conv = 
	//hand-unrolled loop (16 elements for this filter)
	//order changed for increasing the precision
	par.fil[0]*psi_sh[ShBaseElem               ] +
	par.fil[15]*psi_sh[ShBaseElem + 15*NUM_LINES] +
	par.fil[1]*psi_sh[ShBaseElem +   NUM_LINES ] +
	par.fil[14]*psi_sh[ShBaseElem + 14*NUM_LINES] +
	par.fil[2]*psi_sh[ShBaseElem + 2*NUM_LINES ] +
	par.fil[13]*psi_sh[ShBaseElem + 13*NUM_LINES] +
	par.fil[3]*psi_sh[ShBaseElem + 3*NUM_LINES ] +
	par.fil[12]*psi_sh[ShBaseElem + 12*NUM_LINES] +
	par.fil[4]*psi_sh[ShBaseElem + 4*NUM_LINES ] +
	par.fil[11]*psi_sh[ShBaseElem + 11*NUM_LINES] +
	par.fil[5]*psi_sh[ShBaseElem + 5*NUM_LINES ] +
	par.fil[10]*psi_sh[ShBaseElem + 10*NUM_LINES] +
	par.fil[6]*psi_sh[ShBaseElem + 6*NUM_LINES ] +
	par.fil[9]*psi_sh[ShBaseElem + 9*NUM_LINES ] +
	par.fil[7]*psi_sh[ShBaseElem + 7*NUM_LINES ] +
	par.fil[8]*psi_sh[ShBaseElem + 8*NUM_LINES ] ;

      /*
	par.fil[0]*psi_sh[ShBaseElem               ] +
	par.fil[15]*psi_sh[ShBaseElem + 15*par.LinesPerBlock] +
	par.fil[1]*psi_sh[ShBaseElem +   par.LinesPerBlock ] +
	par.fil[14]*psi_sh[ShBaseElem + 14*par.LinesPerBlock] +
	par.fil[2]*psi_sh[ShBaseElem + 2*par.LinesPerBlock ] +
	par.fil[13]*psi_sh[ShBaseElem + 13*par.LinesPerBlock] +
	par.fil[3]*psi_sh[ShBaseElem + 3*par.LinesPerBlock ] +
	par.fil[12]*psi_sh[ShBaseElem + 12*par.LinesPerBlock] +
	par.fil[4]*psi_sh[ShBaseElem + 4*par.LinesPerBlock ] +
	par.fil[11]*psi_sh[ShBaseElem + 11*par.LinesPerBlock] +
	par.fil[5]*psi_sh[ShBaseElem + 5*par.LinesPerBlock ] +
	par.fil[10]*psi_sh[ShBaseElem + 10*par.LinesPerBlock] +
	par.fil[6]*psi_sh[ShBaseElem + 6*par.LinesPerBlock ] +
	par.fil[9]*psi_sh[ShBaseElem + 9*par.LinesPerBlock ] +
	par.fil[7]*psi_sh[ShBaseElem + 7*par.LinesPerBlock ] +
	par.fil[8]*psi_sh[ShBaseElem + 8*par.LinesPerBlock ] ;


	par.fil[0]*psi_sh[ShBaseElem               ] +
	par.fil[1]*psi_sh[ShBaseElem +   par.LinesPerBlock ] +
	par.fil[2]*psi_sh[ShBaseElem + 2*par.LinesPerBlock ] +
	par.fil[3]*psi_sh[ShBaseElem + 3*par.LinesPerBlock ] +
	par.fil[4]*psi_sh[ShBaseElem + 4*par.LinesPerBlock ] +
	par.fil[5]*psi_sh[ShBaseElem + 5*par.LinesPerBlock ] +
	par.fil[6]*psi_sh[ShBaseElem + 6*par.LinesPerBlock ] +
	par.fil[7]*psi_sh[ShBaseElem + 7*par.LinesPerBlock ] +
	par.fil[8]*psi_sh[ShBaseElem + 8*par.LinesPerBlock ] +
	par.fil[9]*psi_sh[ShBaseElem + 9*par.LinesPerBlock ] +
	par.fil[10]*psi_sh[ShBaseElem + 10*par.LinesPerBlock] +
	par.fil[11]*psi_sh[ShBaseElem + 11*par.LinesPerBlock] +
	par.fil[12]*psi_sh[ShBaseElem + 12*par.LinesPerBlock] +
	par.fil[13]*psi_sh[ShBaseElem + 13*par.LinesPerBlock] +
	par.fil[14]*psi_sh[ShBaseElem + 14*par.LinesPerBlock] +
	par.fil[15]*psi_sh[ShBaseElem + 15*par.LinesPerBlock];
      */

      psi_out[BaseElem]=conv;
      //psi_sh[ShBaseElem+par.lowfil*par.LinesPerBlock]; //for testing only

      ShBaseElem += HALF_WARP_SIZE;
      BaseElem += HW_ELEM;
      //BaseElem += par.ElementsPerHalfWarp;

      
    }

 
}

//1D convolution of multiple lines in the same block
//multiplies by the potential and calculate the potential energy
//__global__ void conv1d_stride_pot(int n,int ndat, float *psi_out)
__global__ void conv1d_stride_pot(int n,int ndat, float *psi_in, float *pot, float *psi_out)
{

  //line treated by the given block
  unsigned int lineOffset = min(blockIdx.y*NUM_LINES,ndat-NUM_LINES);
  //starting element treated by the block
  unsigned int elemOffset = min(blockIdx.x*par.ElementsPerBlock,n-par.ElementsPerBlock);

  //line treated by the given block
  //unsigned int lineOffset = min(blockIdx.y*par.LinesPerBlock,ndat-par.LinesPerBlock);
  //starting element treated by the block
  //unsigned int elemOffset = min(blockIdx.x*par.ElementsPerBlock,n-par.ElementsPerBlock);


  //half-warp id
  const unsigned int hwid = threadIdx.y;
  //tid within the HW
  const unsigned int tid_hw = threadIdx.x;

  //shared memory array
  __shared__ float psi_sh[MAX_SHARED_SIZE/sizeof(float)];

  //line treated by the given thread in ndat axis
  //which is the input base element
  unsigned int BaseElem = par.thline[tid_hw] + lineOffset;
  //write data in shared memory
  //element treated by the given thread in n-axis
  unsigned int thelem = par.thelem[tid_hw] + par.hwoffset_copy[hwid];

  unsigned int ShBaseElem = tid_hw + NUM_LINES*par.hwoffset_copy[hwid];
  //unsigned int ShBaseElem = tid_hw + par.LinesPerBlock*par.hwoffset_copy[hwid];

  int epsilon,npos;

  //NOTE: it is assumed that for non-first segments the starting
  //points is far enough for the filter to be contained
  //and the same for non-last segments.
  //in other terms: lenght of the line is always bigger than
  //max(lowfil,lupfil)

  for(int i=0,ipos=elemOffset-par.lowfil+thelem;i < par.hwelem_copy[hwid] ; ++i)
    {
      //control flag for periodic boundary conditions
      epsilon=(ipos < 0 ? -1 : ipos/n);
      npos=ipos-epsilon*n;

      psi_sh[ShBaseElem]=psi_in[BaseElem+ndat*npos];
      //psi_sh[ShBaseElem]=tex1Dfetch(psi_tex,BaseElem+ndat*npos);

      ShBaseElem += HALF_WARP_SIZE;
      ipos += HW_ELEM;
      //ipos += par.ElementsPerHalfWarp;
      
    }

  //end shared memory copy
  __syncthreads();

  //element treated by the given thread in n-axis
  thelem = par.thelem[tid_hw] + par.hwoffset_calc[hwid];
  //base element for the given thread in shared memory
  ShBaseElem = tid_hw + NUM_LINES*par.hwoffset_calc[hwid];
  //ShBaseElem = tid_hw + par.LinesPerBlock*par.hwoffset_calc[hwid];

  //output base element, from the input one
  BaseElem =  n*BaseElem+ thelem + elemOffset;

  //limit element for which the block treats unique elements

  //perform convolution in shared memory 
  //each thread calculate a number of elements, identical for each
  //half-warp
  //#pragma unroll 5 (to be tested if it is important)

  /* suspend the potential energy calculation due to doubling of
     addresses
     perhaps a ddot strategy has better performances
  //per thread value of the potential energy
  __shared__ float epot_th[16][HALF_WARP_SIZE];
  //initalize suitable value
  epot_th[hwid][tid_hw]=0.f;
  */

  for(int i=0;i < par.hwelem_calc[hwid]; ++i)
    {
      //values of the convolution
      register float conv = 
	//hand-unrolled loop (16 elements for this filter)
	//order changed for increasing the precision
	par.fil[0]*psi_sh[ShBaseElem               ] +
	par.fil[15]*psi_sh[ShBaseElem + 15*NUM_LINES] +
	par.fil[1]*psi_sh[ShBaseElem +   NUM_LINES ] +
	par.fil[14]*psi_sh[ShBaseElem + 14*NUM_LINES] +
	par.fil[2]*psi_sh[ShBaseElem + 2*NUM_LINES ] +
	par.fil[13]*psi_sh[ShBaseElem + 13*NUM_LINES] +
	par.fil[3]*psi_sh[ShBaseElem + 3*NUM_LINES ] +
	par.fil[12]*psi_sh[ShBaseElem + 12*NUM_LINES] +
	par.fil[4]*psi_sh[ShBaseElem + 4*NUM_LINES ] +
	par.fil[11]*psi_sh[ShBaseElem + 11*NUM_LINES] +
	par.fil[5]*psi_sh[ShBaseElem + 5*NUM_LINES ] +
	par.fil[10]*psi_sh[ShBaseElem + 10*NUM_LINES] +
	par.fil[6]*psi_sh[ShBaseElem + 6*NUM_LINES ] +
	par.fil[9]*psi_sh[ShBaseElem + 9*NUM_LINES ] +
	par.fil[7]*psi_sh[ShBaseElem + 7*NUM_LINES ] +
	par.fil[8]*psi_sh[ShBaseElem + 8*NUM_LINES ] ;

      /*
	par.fil[0]*psi_sh[ShBaseElem               ] +
	par.fil[15]*psi_sh[ShBaseElem + 15*par.LinesPerBlock] +
	par.fil[1]*psi_sh[ShBaseElem +   par.LinesPerBlock ] +
	par.fil[14]*psi_sh[ShBaseElem + 14*par.LinesPerBlock] +
	par.fil[2]*psi_sh[ShBaseElem + 2*par.LinesPerBlock ] +
	par.fil[13]*psi_sh[ShBaseElem + 13*par.LinesPerBlock] +
	par.fil[3]*psi_sh[ShBaseElem + 3*par.LinesPerBlock ] +
	par.fil[12]*psi_sh[ShBaseElem + 12*par.LinesPerBlock] +
	par.fil[4]*psi_sh[ShBaseElem + 4*par.LinesPerBlock ] +
	par.fil[11]*psi_sh[ShBaseElem + 11*par.LinesPerBlock] +
	par.fil[5]*psi_sh[ShBaseElem + 5*par.LinesPerBlock ] +
	par.fil[10]*psi_sh[ShBaseElem + 10*par.LinesPerBlock] +
	par.fil[6]*psi_sh[ShBaseElem + 6*par.LinesPerBlock ] +
	par.fil[9]*psi_sh[ShBaseElem + 9*par.LinesPerBlock ] +
	par.fil[7]*psi_sh[ShBaseElem + 7*par.LinesPerBlock ] +
	par.fil[8]*psi_sh[ShBaseElem + 8*par.LinesPerBlock ] ;

	par.fil[0]*psi_sh[ShBaseElem               ] +
	par.fil[1]*psi_sh[ShBaseElem +   par.LinesPerBlock ] +
	par.fil[2]*psi_sh[ShBaseElem + 2*par.LinesPerBlock ] +
	par.fil[3]*psi_sh[ShBaseElem + 3*par.LinesPerBlock ] +
	par.fil[4]*psi_sh[ShBaseElem + 4*par.LinesPerBlock ] +
	par.fil[5]*psi_sh[ShBaseElem + 5*par.LinesPerBlock ] +
	par.fil[6]*psi_sh[ShBaseElem + 6*par.LinesPerBlock ] +
	par.fil[7]*psi_sh[ShBaseElem + 7*par.LinesPerBlock ] +
	par.fil[8]*psi_sh[ShBaseElem + 8*par.LinesPerBlock ] +
	par.fil[9]*psi_sh[ShBaseElem + 9*par.LinesPerBlock ] +
	par.fil[10]*psi_sh[ShBaseElem + 10*par.LinesPerBlock] +
	par.fil[11]*psi_sh[ShBaseElem + 11*par.LinesPerBlock] +
	par.fil[12]*psi_sh[ShBaseElem + 12*par.LinesPerBlock] +
	par.fil[13]*psi_sh[ShBaseElem + 13*par.LinesPerBlock] +
	par.fil[14]*psi_sh[ShBaseElem + 14*par.LinesPerBlock] +
	par.fil[15]*psi_sh[ShBaseElem + 15*par.LinesPerBlock];
      */

      //register float v=tex1Dfetch(pot_tex,BaseElem);

      psi_out[BaseElem]=conv*pot[BaseElem];

      //update potential energy
      //not efficient calculation, update the energy only if the element
      //treated is unique
      //epot_th[hwid][tid_hw] += conv*v*conv;
      

      ShBaseElem += HALF_WARP_SIZE;
      BaseElem += HW_ELEM;
      //BaseElem += par.ElementsPerHalfWarp;

      
    }


  /* partial reduction of the potential energy.
     not valid due to duplication of calculations
  //here we should add the reduction procedure for a given subset of
  //elements. each block will provide only one value and copy it on
  //global memory

  //wait until each thread has finished
  __syncthreads();

  //now reduce by knowing that the blockSize is always less or equal
  //than 256
  //use the rationale of parallel reduction indicated in CUDA examples
  if (blockDim.y >= 16)
    {
      if (hwid < 8){epot_th[hwid][tid_hw]+=epot_th[hwid+8][tid_hw];}
      __syncthreads();
    }
  if (blockDim.y >= 8)
    {
      if (hwid < 4){epot_th[hwid][tid_hw]+=epot_th[hwid+4][tid_hw];}
      __syncthreads();
    }
  //then add the statements which do not need to syncthreads

  */
  

 
}

//interface, only the 1d convolution
extern "C" 
void g1dconv_(int *n, 
	      int *ndat, 
	      float **data_in, 
	      float **data_out, 
	      float *filters, 
	      int *lowfil, 
	      int *lupfil)
{

  const int n1 = *ndat;
  const int n2 = *n+1;

  
  if(dogenconv(n1,
	       n2, 
	       *data_in,
	       *data_out,
	       *lowfil,
	       *lupfil) != 0)
    {
      return;
    } 
  return; 
}


int dogenconv(int ndat,
	      int n, 
	      float *GPU_idata,
	      float *GPU_odata,
	      int lowfil,
	      int lupfil)
{

  //create the parameters
  par_t parCPU;

  //calculate the number of threads and blocks
  //unsigned int num_lines = min(16,ndat); //hard coded for the moment
  unsigned int numBlocks,linecuts,num_halfwarps;
  //int tex_offset;
  //size_t offset;

  constantParameters(&parCPU,&num_halfwarps,n,ndat,lowfil,lupfil,
		     &linecuts,&numBlocks);
  //constantParameters(&parCPU,&num_halfwarps,num_lines,n,ndat,lowfil,lupfil,
  //&linecuts,&numBlocks);



  //printf("num_blocksx %i, num_blocksy %i, halfwarps %i\n",linecuts,numBlocks,num_halfwarps);

  //send them to constant memory
  if(hipMemcpyToSymbol(HIP_SYMBOL(par),&parCPU, sizeof(par_t)) != 0)
    {
      printf("MemcpyToSymbol error\n");

      return 1;
    }
 
  //define the number of threads and blocks according to parameter definitions
  dim3  grid1(linecuts,  numBlocks, 1);  
  dim3  threads1(HALF_WARP_SIZE, num_halfwarps , 1);

  //bind the texture reference to the input array
  //hipBindTexture(NULL,psi_tex,GPU_idata,n*ndat*sizeof(float));

  //element offset for reading from the texture
  //tex_offset = offset/sizeof(float);
  
  //printf(" offset %i\n",tex_offset); 
  //launch the kernel grid
  //conv1d_stride <<< grid1, threads1 >>>(n,ndat, GPU_odata);
  conv1d_stride <<< grid1, threads1 >>>(n,ndat, GPU_idata, GPU_odata);

  //unbind the texture
  //hipUnbindTexture(psi_tex);

  hipDeviceSynchronize();

  return 0;

}

/****/
