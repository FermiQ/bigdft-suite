#include <iostream>
#include <stdio.h>
#include "exceptions.h"  

#include <config.h>

//functions for allocate / deallocate / send memory to GPU


#define CUERR { hipError_t err; \
 if ((err = hipGetLastError()) != hipSuccess) { \
 printf("CUDA error: %s, line %d\n", hipGetErrorString(err), __LINE__); }}




extern "C" 
void FC_FUNC(sg_cpu_pinned_alloc,SG_CPU_PINNED_ALLOC)(void **CPU_pointer,
			   int *nsize, //memory size
			   int *precision,
			   int *ierr) // error code, 1 if failure
{
  unsigned int mem_size = (*nsize)*(*precision);
  *ierr=0;
  
  try
    {
      check<cuda_error>(hipHostMalloc(CPU_pointer, mem_size ) != hipSuccess,"CPU pinned allocation",__FILE__,__LINE__);
    }
  
  catch(std::exception &e)
    {
      
      std::cerr << "*** CUDA ERROR DETECTED" << std::endl;
      std::cerr << "ERROR MESSAGE : " << e.what() << std::endl;
      *ierr = 1;
    }
 
}


extern "C" 
void FC_FUNC(sg_cpu_pinned_free,SG_CPU_PINNED_FREE)(void **CPU_pointer,
			  int *ierr) // error code, 1 if failure
{
  *ierr=0;

  try
    {
      check<cuda_error>(hipHostFree(*CPU_pointer) != hipSuccess,"hipHostFree",__FILE__,__LINE__);
    }
  
  catch(std::exception &e)
    {
      
      std::cerr << "*** CUDA ERROR DETECTED" << std::endl;
      std::cerr << "ERROR MESSAGE : " << e.what() << std::endl;
      *ierr = 1;
    }


  CUERR;
}






extern "C" 
void FC_FUNC(sg_gpu_alloc,SG_GPU_ALLOC)(void **GPU_pointer, // pointer indicating the GPU address
		    int *nsize, //memory size
		    int *precision,
		    int *ierr) // error code, 1 if failure
{

  unsigned int mem_size = (*nsize)*(*precision);

  
  *ierr=0;


 try
    {
      check<cuda_error>(hipMalloc( GPU_pointer, mem_size) != 0,"GPU allocation",__FILE__,__LINE__);
    }
  
 catch(std::exception &e)
    {
      
      std::cerr << "*** CUDA ERROR DETECTED" << std::endl;
      std::cerr << "ERROR MESSAGE : " << e.what() << std::endl;
      *ierr = 1;
    }
}


extern "C" 
void FC_FUNC(sg_gpu_free,SG_GPU_FREE)(void **GPU_pointer, // pointer indicating the GPU address
		   int *ierr) // error code, 1 if failure
{

  *ierr=0;

 try
    {
      check<cuda_error>(hipFree(*GPU_pointer) != 0,"CUDA free",__FILE__,__LINE__);
    }
  
  catch(std::exception e)
    {
      
      std::cerr << "*** CUDA ERROR DETECTED" << std::endl;
      std::cerr << "ERROR MESSAGE : " << e.what() << std::endl;
      *ierr = 1;
    }
}



extern "C"
void FC_FUNC(sg_gpu_imm_send,SG_GPU_IMM_SEND)(void **GPU_pointer,
		       void *CPU_pointer, 
		       int *nsize,
		       int *precision,
		       int *ierr)
{
  unsigned int mem_size = (*nsize)*(*precision);


  *ierr=0;

  try
    {
      check<cuda_error>(hipMemcpy(*GPU_pointer, CPU_pointer, mem_size, hipMemcpyHostToDevice)  != 0,"copy host to device",__FILE__,__LINE__);
    }
  
  catch(std::exception &e)
    {
      
      std::cerr << "*** CUDA ERROR DETECTED" << std::endl;
      std::cerr << "ERROR MESSAGE : " << e.what() << std::endl;
      *ierr = 1;
    }

}



extern "C" 
void FC_FUNC(sg_gpu_imm_recv,SG_GPU_IMM_RECV)(void *CPU_pointer, 
		       void **GPU_pointer,
		       int *nsize,
		       int *precision,
		       int *ierr)
{

  unsigned int mem_size = (*nsize)*(*precision);

 
  *ierr=0;


try
    {
      check<cuda_error>(hipMemcpy(CPU_pointer,*GPU_pointer, mem_size, hipMemcpyDeviceToHost),"copy device to host",__FILE__,__LINE__);
    }
  
  catch(std::exception& e)
    {
      
      std::cerr << "*** CUDA ERROR DETECTED" << std::endl;
      std::cerr << "ERROR MESSAGE : " << e.what() << std::endl;
      *ierr = 1;
    }



}


