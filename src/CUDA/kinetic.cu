#include "hip/hip_runtime.h"
/****u* CUDA/kinetic.cu
**
** 
** AUTHOR
**  Luigi Genovese
**
** SOURCE
*/
  
#include <stdio.h>

#include "kinetic.h"

#include "commonDef.h"


#include "reduction.h"


#include "structDef_kinetic.h"



__constant__ parGPU_t par[3];


#include "kernels_kinetic.hcu"

// Kinetic parameters to be used for calculating the convolution
template<typename T>
void KParameters(parGPU_t* par,
		 unsigned int* num_halfwarps,
		 int n,
		 int ndat,
		 unsigned int* linecuts,
		 unsigned int* num_blocks)

{

  //number of total allowed elements of a input line
   int num_elem_tot = MAX_SHARED_SIZE/sizeof(T)/NUM_LINES; //between1024and64
  
  //number of elements of the output
   int num_elem_max = min(num_elem_tot-LOWFILK-LUPFILK-1,n); //between 996 and 35 

  //number of pieces in which a line is divided
  //if the line is too small and not a multiple of ElementsPerHalfWarp
  //divide the line in two
  *linecuts=
    (n <= num_elem_max && n % HW_ELEM !=0 ? 2 : (n-1)/num_elem_max+1);

  //number of blocks in ndat direction
  *num_blocks=((ndat-1)/NUM_LINES + 1);

  //printf("num_elem_tot %i,num_elem_max %i,linecuts %i,num_blocks %i,elemperHW %i \n",
  //num_elem_tot,num_elem_max,*linecuts,*num_blocks, par -> ElementsPerHalfWarp);

  //number of elements treated by each block 
  //this may pose problems for values of n dimensions less than 48
  //when n is not a multiple of ElementsPerHalfWarp
  par->ElementsPerBlock = 
    min(HW_ELEM*(((n-1)/(int)(*linecuts))/HW_ELEM+1),n);

  int halfwarps=16;
  //calculate the maximum number of halfwarps (between 4 and 16)
  for(int i =3; i>=0; --i)
    {
      if(par->ElementsPerBlock/HW_ELEM >= 1 << i)
	{
	  halfwarps = 1 << i;
	  break;
	}
    }

  *num_halfwarps = halfwarps;

  for(int j=0;j < HALF_WARP_SIZE ; ++j)
    {
      par->thline[j]= j & (NUM_LINES - 1); //num_lines always a power of two 
      par->thelem[j]= j / NUM_LINES; 
    }

  //define the sequences of the number of elements
  correctSequence(halfwarps,par->ElementsPerBlock/HW_ELEM,par->hwelem_calc);

  correctSequence(halfwarps,(par->ElementsPerBlock+LOWFILK+LUPFILK+1)/HW_ELEM,
		  par->hwelem_copy);

  //define the offsets
  for(int j=0,pos_calc=0,pos_copy=0;j < halfwarps ; ++j)
    {
      par->hwoffset_calc[j]=pos_calc;
      par->hwoffset_copy[j]=pos_copy;
      pos_calc+=HW_ELEM*par->hwelem_calc[j];
      pos_copy+=HW_ELEM*par->hwelem_copy[j];
    }
 
}



template<typename T>
int kineticfilter(int n1,int n2, int n3,
		  T h1,T h2,T h3,T c,
		  T *x,
		  T *workx,
		  T *y,
		  T *worky,
		  T *ekin)
{

  //create the parameters
  parGPU_t parCPU[3];

  //calculate the number of threads and blocks
  unsigned int numBlocks,linecuts,num_halfwarps;

  //calculate the parameters in constant memory for each of the 1D convolution
  //define the number of threads and blocks according to parameter definitions
  KParameters<T>(&parCPU[2],&num_halfwarps,n3,n1*n2,&linecuts,&numBlocks);
  dim3  grid3(linecuts,  numBlocks, 1);  
  dim3  threads3(HALF_WARP_SIZE, num_halfwarps , 1);

  //printf("num_blocksx %i, num_blocksy %i, halfwarps %i,n1,ndat, %i %i\n",
  //linecuts,numBlocks,num_halfwarps,n3,n1*n2);

  KParameters<T>(&parCPU[1],&num_halfwarps,n2,n1*n3,&linecuts,&numBlocks);
  dim3  grid2(linecuts,  numBlocks, 1);  
  dim3  threads2(HALF_WARP_SIZE, num_halfwarps , 1);

  //printf("num_blocksx %i, num_blocksy %i, halfwarps %i,n1,ndat, %i %i\n",
  //linecuts,numBlocks,num_halfwarps,n2,n1*n3);

  KParameters<T>(&parCPU[0],&num_halfwarps,n1,n2*n3,&linecuts,&numBlocks);
  dim3  grid1(linecuts,  numBlocks, 1);  
  dim3  threads1(HALF_WARP_SIZE, num_halfwarps , 1);

  //printf("num_blocksx %i, num_blocksy %i, halfwarps %i,n1,ndat, %i %i\n",
  //linecuts,numBlocks,num_halfwarps,n1,n3*n2);

  //send them to constant memory, once and for all
  if(hipMemcpyToSymbol(HIP_SYMBOL(*par),&parCPU, 3*sizeof(parGPU_t)) != 0)
    {
      printf("MemcpyToSymbol error\n");

      return 1;
    }


  //here the worky array should be initialised to c*x
  c_initialize<T> <<< grid3, threads3 >>>(n3,n1*n2,x,worky,c,2);
  hipDeviceSynchronize();


  //define the scale factor to be applied to the convolution
  T scale=0.5/(h3*h3);

  kinetic1d<T> <<< grid3, threads3 >>>(n3,n1*n2,scale,x,workx,worky,y,2);
  hipDeviceSynchronize();

  scale=0.5/(h2*h2);
  kinetic1d<T> <<< grid2, threads2 >>>(n2,n1*n3,scale,workx,x,y,worky,1);
  hipDeviceSynchronize();

  scale=0.5/(h1*h1);
  kinetic1d<T> <<< grid1, threads1 >>>(n1,n2*n3,scale,x,workx,worky,y,0);
  hipDeviceSynchronize();

  //then calculate the kinetic energy
  reducearrays<T>(n1,n2*n3,x,y,ekin);
  hipDeviceSynchronize();

  return 0;

}

template<typename T>
int k1d(int ndat, int n,
	T h,T c,
	T *x,
	T *workx,
	T *y,
	T *worky,
	T *ekin)
{

  //create the parameters
  parGPU_t parCPU[1];

  //calculate the number of threads and blocks
  unsigned int numBlocks,linecuts,num_halfwarps;

  //calculate the parameters in constant memory for each of the 1D convolution
  //define the number of threads and blocks according to parameter definitions
  KParameters<T>(&parCPU[0],&num_halfwarps,n,ndat,&linecuts,&numBlocks);
  dim3  grid3(linecuts,  numBlocks, 1);  
  dim3  threads3(HALF_WARP_SIZE, num_halfwarps , 1);

  //send them to constant memory, once and for all
  if(hipMemcpyToSymbol(HIP_SYMBOL(*par),&parCPU, sizeof(parGPU_t)) != 0)
    {
      printf("MemcpyToSymbol error\n");

      return 1;
    }


  //define the scale factor to be applied to the convolution
  T scale=0.5/(h*h);

  //here the worky array should be initialised to c*x
  c_initialize<T> <<< grid3, threads3 >>>(n,ndat,x,worky,c,0);
  hipDeviceSynchronize();

  kinetic1d<T> <<< grid3, threads3 >>>(n,ndat,scale,x,workx,worky,y,0);
  hipDeviceSynchronize();

  //then calculate the kinetic energy
  reducearrays<T>(n,ndat,x,y,ekin);
  hipDeviceSynchronize();
  return 0;

}

/****/



extern "C" 
void kineticterm_(int *n1,int *n2,int *n3,
		  float *hx,float *hy,float *hz,float *c,
		  float **x,float **y,float **workx,float **worky,
		  float *ekin) 

{

  
  if(kineticfilter<float>(*n1+1,*n2+1,*n3+1,
			  *hx,*hy,*hz,*c,
			  *x,*workx,*y,*worky,
			  ekin) != 0)
    {
      printf("ERROR: GPU kineticfilter\n ");
      return;
    } 
  return; 
}

extern "C" 
void kinetictermd_(int *n1,int *n2,int *n3,
		   double *hx,double *hy,double *hz,double *c,
		   double **x,double **y,double **workx,double **worky,
		   double *ekin) 

{

  
  if(kineticfilter<double>(*n1+1,*n2+1,*n3+1,
			  *hx,*hy,*hz,*c,
			  *x,*workx,*y,*worky,
			  ekin) != 0)
    {
      printf("ERROR: GPU kineticfilter\n ");
      return;
    } 
  return; 
  }

extern "C" 
void kinetic1d_(int *n,int *ndat,
		double *h,double *c,
		double **x,double **y,double **workx,double **worky,
		double *ekin) 

{

  
  if(k1d<double>(*ndat,*n+1,
		 *h,*c,
		 *x,*workx,*y,*worky,ekin) != 0)
    {
      printf("ERROR: GPU kineticfilter\n ");
      return;
    } 
  return; 
  }
