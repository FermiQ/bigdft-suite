#include "hip/hip_runtime.h"
/****u* CUDA/kinetic.cu
**
** 
** AUTHOR
**  Luigi Genovese
**
** SOURCE
*/
  
#include <stdio.h>

#include "kinetic.h"

#include "commonDef.h"


#include "reduction.h"


#include "structDef_kinetic.h"



__constant__ parGPU_t par[3];


#include "kernels_kinetic.hcu"

// Kinetic parameters to be used for calculating the convolution
/*
template<typename T>
void KParameters(parGPU_t* par,
		 unsigned int* num_halfwarps,
		 int n,
		 int ndat,
		 unsigned int* linecuts,
		 unsigned int* num_blocks)

{

  //number of total allowed elements of a input line
   int num_elem_tot = MAX_SHARED_SIZE/sizeof(T)/NUM_LINES; //between1024and64
  
  //number of elements of the output
   int num_elem_max = min(num_elem_tot-LOWFILK-LUPFILK-1,n); //between 996 and 35 

  //number of pieces in which a line is divided
  //if the line is too small and not a multiple of ElementsPerHalfWarp
  //divide the line in two
  *linecuts=
    (n <= num_elem_max && n % HW_ELEM !=0 ? 2 : (n-1)/num_elem_max+1);

  //number of blocks in ndat direction
  *num_blocks=((ndat-1)/NUM_LINES + 1);

  //printf("num_elem_tot %i,num_elem_max %i,linecuts %i,num_blocks %i,elemperHW %i \n",
  //num_elem_tot,num_elem_max,*linecuts,*num_blocks, par -> ElementsPerHalfWarp);

  //number of elements treated by each block 
  //this may pose problems for values of n dimensions less than 48
  //when n is not a multiple of ElementsPerHalfWarp
  par->ElementsPerBlock = 
    min(HW_ELEM*(((n-1)/(int)(*linecuts))/HW_ELEM+1),n);

  int halfwarps=16;
  //calculate the maximum number of halfwarps (between 4 and 16)
  for(int i =3; i>=0; --i)
    {
      if(par->ElementsPerBlock/HW_ELEM >= 1 << i)
	{
	  halfwarps = 1 << i;
	  break;
	}
    }

  *num_halfwarps = halfwarps;

  for(int j=0;j < HALF_WARP_SIZE ; ++j)
    {
      par->thline[j]= j & (NUM_LINES - 1); //num_lines always a power of two 
      par->thelem[j]= j / NUM_LINES; 
    }

  //define the sequences of the number of elements
  correctSequence(halfwarps,par->ElementsPerBlock/HW_ELEM,par->hwelem_calc);

  correctSequence(halfwarps,(par->ElementsPerBlock+LOWFILK+LUPFILK+1)/HW_ELEM,
		  par->hwelem_copy);

  //define the offsets
  for(int j=0,pos_calc=0,pos_copy=0;j < halfwarps ; ++j)
    {
      par->hwoffset_calc[j]=pos_calc;
      par->hwoffset_copy[j]=pos_copy;
      pos_calc+=HW_ELEM*par->hwelem_calc[j];
      pos_copy+=HW_ELEM*par->hwelem_copy[j];
    }
 
}
*/

extern "C" 
void kineticterm_(int *n1,int *n2,int *n3,
		  float *hx,float *hy,float *hz,float *c,
		  float **x,float **y,float **workx,float **worky,
		  float *ekin) 

{

  
  if(kineticfilter<float>(*n1+1,*n2+1,*n3+1,
			  *hx,*hy,*hz,*c,
			  *x,*workx,*y,*worky,
			  ekin) != 0)
    {
      printf("ERROR: GPU kineticfilter\n ");
      return;
    } 
  return; 
}

extern "C" 
void kinetictermd_(int *n1,int *n2,int *n3,
		   double *hx,double *hy,double *hz,double *c,
		   double **x,double **y,double **workx,double **worky,
		   double *ekin) 

{

  
  if(kineticfilter<double>(*n1+1,*n2+1,*n3+1,
			  *hx,*hy,*hz,*c,
			  *x,*workx,*y,*worky,
			  ekin) != 0)
    {
      printf("ERROR: GPU kineticfilter\n ");
      return;
    } 
  return; 
  }

extern "C" 
void kinetic1d_(int *n,int *ndat,
		double *h,double *c,
		double **x,double **y,double **workx,double **worky,
		double *ekin) 

{

  
  if(k1d<double>(*ndat,*n+1,
		 *h,*c,
		 *x,*workx,*y,*worky,ekin) != 0)
    {
      printf("ERROR: GPU kineticfilter\n ");
      return;
    } 
  return; 
}
