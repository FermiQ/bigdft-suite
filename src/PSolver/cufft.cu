#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include "hipfft/hipfft.h"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"
 
#define DOUBLE

#ifdef DOUBLE
#define Complex  hipfftDoubleComplex
#define Real double
#define Transform HIPFFT_Z2Z
#define TransformExec hipfftExecZ2Z
#else
#define Complex  hipfftComplex
#define Real float
#define Transform HIPFFT_C2C
#define TransformExec hipfftExecC2C
#endif

#define TILE_DIM  8

// synchronize blocks
extern "C" void synchronize_() {
 
  hipDeviceSynchronize();
}

// allocate device memory
extern "C" void cudamalloc_(int *size, Real **d_data) {

  hipMalloc((void**)d_data, sizeof(Real)*(*size));
  if( hipGetLastError() != hipSuccess)
      printf("allocate error\n");
}

extern "C" void cudafree_(Real **d_data) {

  hipFree(*d_data);
}

// set device memory
extern "C" void reset_gpu_data_(int *size, Real* h_data, Real **d_data){

 hipMemcpy(*d_data, h_data, sizeof(Real)*(*size),
         hipMemcpyHostToDevice);
 if( hipGetLastError() != hipSuccess)
      printf("transfer error\n");

}

// read device memory
extern "C" void get_gpu_data_(int *size, Real *h_data, Real **d_data) {

 hipMemcpy(h_data, *d_data, sizeof(Real)*(*size),
         hipMemcpyDeviceToHost);
 if (hipGetLastError() != hipSuccess)
        printf("transfer back error\n");
}


// transpose
__global__ void transpose(Complex *idata, Complex *odata,
        int width, int height)
{
  __shared__ Complex tile[TILE_DIM][TILE_DIM+1];

  int xIndex = blockIdx.x * TILE_DIM + threadIdx.x;
  int yIndex = blockIdx.y * TILE_DIM + threadIdx.y;
  int index_in = xIndex + (yIndex)*(width);
  int xIndex1 = blockIdx.y * TILE_DIM + threadIdx.x;
  int yIndex1 = blockIdx.x * TILE_DIM + threadIdx.y;
  int index_out = xIndex1 + (yIndex1)*height;

  if (xIndex < width && yIndex < height)
      tile[threadIdx.y][threadIdx.x] = idata[index_in];
    __syncthreads();

  if (xIndex1 < height && yIndex1 < width) {
      odata[index_out] = tile[threadIdx.x][threadIdx.y];
  }
}

// transpose together with spread operation
__global__ void transpose_spread(Complex *idata, Complex *odata, 
	int width, int height, int bign_h)
{
  __shared__ Complex tile[TILE_DIM][TILE_DIM+1];

  int xIndex = blockIdx.x * TILE_DIM + threadIdx.x;
  int yIndex = blockIdx.y * TILE_DIM + threadIdx.y;
  int index_in = xIndex + (yIndex)*(width);
  int xIndex1 = blockIdx.y * TILE_DIM + threadIdx.x;
  int yIndex1 = blockIdx.x * TILE_DIM + threadIdx.y;
  int index_out = xIndex1 + (yIndex1)*height;
  int div = index_out / bign_h;
  int mod = index_out % bign_h;
  index_out = div * (bign_h << 1) + mod+bign_h;
  int plus = -bign_h;

  if (xIndex < width && yIndex < height)
      tile[threadIdx.y][threadIdx.x] = idata[index_in];
    __syncthreads();

  if (xIndex1 < height && yIndex1 < width) {
      odata[index_out] = tile[threadIdx.x][threadIdx.y];
    #ifdef DOUBLE
      odata[index_out + plus] = make_double2(0., 0.);
    #else
      odata[index_out + plus] = make_float2(0.f, 0.f);
    #endif
  }
}

// transpose together with inverse spread operation
__global__ void transpose_spread_i(Complex *idata, Complex *odata,
        int width, int height, int bign_h)
{
  __shared__ Complex tile[TILE_DIM][TILE_DIM+1];

  int xIndex = blockIdx.x * TILE_DIM + threadIdx.x;
  int yIndex = blockIdx.y * TILE_DIM + threadIdx.y;
  int index_in = xIndex + (yIndex)*(width);
  int xIndex1 = blockIdx.y * TILE_DIM + threadIdx.x;
  int yIndex1 = blockIdx.x * TILE_DIM + threadIdx.y;
  int index_out = xIndex1 + (yIndex1)*height;
  int div = index_in / bign_h;
  int mod = index_in % bign_h;
  index_in = div * (bign_h << 1) + mod;

  if (xIndex < width && yIndex < height)
      tile[threadIdx.y][threadIdx.x] = idata[index_in];
    __syncthreads();

  if (xIndex1 < height && yIndex1 < width)
      odata[index_out] = tile[threadIdx.x][threadIdx.y];
}

// spread operation
__global__ void spread(Real* src, unsigned int spitch, Real* dst, unsigned int dpitch)
{
   unsigned int bid = blockIdx.y * gridDim.x + blockIdx.x;
   unsigned int tid = threadIdx.x;
 
   Real res = (tid >= spitch) ? src[bid * spitch + tid-spitch] : 0.0;
   if( tid < dpitch) {
	dst[bid * dpitch + tid] = res;
   }
}

// inverse spread operation
__global__ void spread_i(Real* src, unsigned int spitch, Real* dst, unsigned int dpitch)
{
   unsigned int bid = blockIdx.y * gridDim.x + blockIdx.x;
   unsigned int tid = threadIdx.x;

   Real res = src[bid * dpitch + tid];
   if( tid < dpitch) dst[bid * spitch + tid] = res;
}

// spread operation for 2nd dim
__global__ void spread_y(Complex* src, Complex* dst)
{
   unsigned int tid = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x + threadIdx.x;
   unsigned int tid1 = (blockIdx.y * gridDim.x * 2 + blockIdx.x) * blockDim.x + threadIdx.x;

   Complex res =  src[tid];
   dst[tid1 + blockDim.x*gridDim.x] = res;
   int plus = -gridDim.x;
#ifdef DOUBLE
   dst[tid1 + blockDim.x*(gridDim.x + plus)] = make_double2(0., 0.);
#else
   dst[tid1 + blockDim.x*(gridDim.x + plus)] = make_float2(0.f, 0.f);
#endif
}

// inverse spread operation for 2nd dim
__global__ void spread_y_i(Complex* src, Complex* dst)
{
   unsigned int tid = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x + threadIdx.x;
   unsigned int tid1 = (blockIdx.y * gridDim.x * 2 + blockIdx.x) * blockDim.x + threadIdx.x;

   Complex res =  src[tid1];
   dst[tid] = res;
}

// multiply with potential
__global__ void multiply_kernel(int nx, int ny, int nz, Complex *d_data, Real *d_kernel, Real scal) {

 int tj = threadIdx.x;
 int td = blockDim.x;

 int blockData = (nx*ny*nz)/(gridDim.x*gridDim.y);

 int jj = (blockIdx.y*gridDim.x + blockIdx.x)*blockData;

 for (int k=0; k<blockData/td; k++) {
     d_data[jj + tj+ k*td].x *= d_kernel[jj + tj+ k*td]*scal;
     d_data[jj + tj+ k*td].y *= d_kernel[jj + tj+ k*td]*scal;
 }

}

// scale
__global__ void scale_kernel(int nx, int ny, int nz, Complex *d_data, Real mult) {

 int tj = threadIdx.x;
 int td = blockDim.x;

 int blockData = (nx*ny*nz)/(gridDim.x*gridDim.y);

 int jj = (blockIdx.y*gridDim.x + blockIdx.x)*blockData;

 for (int k=0; k<blockData/td; k++) {
     d_data[jj + tj+ k*td].x *= mult;
     d_data[jj + tj+ k*td].y *= mult;
 }

}

__global__ void zero(int nx, int ny, int nz, Real *z) {

        int tj = threadIdx.x;
        int td = blockDim.x;

	int blockData = (nx*ny*nz)/(gridDim.x*gridDim.y);

        int jj = ((blockIdx.y)*gridDim.x + (blockIdx.x))*blockData;

        for (int k=0; k<blockData/td; k++) {
        	z[jj + tj+ k*td] = 0.0;
        }
}

__global__ void copy_0(int nx, int ny, int nz, Real *in, Real *out) {

	int tj = threadIdx.x;
        int td = blockDim.x;

        int jj =  (blockIdx.y*nx*ny/4 + blockIdx.x*nx/2);
        int jj1 =  ((blockIdx.y+nz/2)*nx*ny + (blockIdx.x+ny/2)*nx);


        out[jj1+tj+td] = in[jj+tj];

}

__global__ void copy(int nx,int ny,int nz, Real *in, Real *out) {

        int tj = threadIdx.x;
        //int td = blockDim.x;

        int jj =  (blockIdx.y*nx*ny/4 + blockIdx.x*nx/2);
        int jj1 =  ((blockIdx.y)*nx*ny + (blockIdx.x)*nx);

        out[jj+tj] = in[jj1+tj];
}

/************ 1D transform *************/

extern "C" void cuda_1d_plan_(int *NX_p, int *Nbatch_p,
                 hipfftHandle *plan) {

 int NX = *NX_p;
 int Nbatch = *Nbatch_p;

 int n1d[3]= {NX, 1, 1};

 if(hipfftPlanMany(plan,  1, n1d,
              NULL, 1, NX,
              NULL, 1, NX, Transform, Nbatch) != HIPFFT_SUCCESS)
      printf("Error creating plan\n");

 //hipfftPlan1d(plan, NX, Transform, Nbatch );

}

extern "C" void cuda_1d_forward_(hipfftHandle *plan,
                Complex **d_data, Complex **d_data2) {

   if( TransformExec(*plan, *d_data, *d_data2, HIPFFT_FORWARD)!= HIPFFT_SUCCESS){
      printf("error in 1D forward transform\n");
   }

}

extern "C" void cuda_1d_inverse_(hipfftHandle *plan,
                Complex **d_data, Complex **d_data2) {

   if( TransformExec(*plan, *d_data, *d_data2, HIPFFT_BACKWARD)!= HIPFFT_SUCCESS){
      printf("error in 1D inverse transform\n");
   }

}

/************ 2D transform *************/

extern "C" void cuda_2d_plan_(int *NX_p, int *NY_p, int *Nbatch_p,
                 hipfftHandle *plan) {

 int NX = *NX_p;
 int NY = *NY_p;
 int Nbatch = *Nbatch_p;

 int n1d[3]= {NX, NY, 1};

 if(hipfftPlanMany(plan,  1, n1d,
              NULL, 1, NX*NY,
              NULL, 1, NX*NY, Transform, Nbatch) != HIPFFT_SUCCESS)
      printf("Error creating plan\n");

}

extern "C" void cuda_2d_forward_(hipfftHandle *plan,
                Complex **d_data, Complex **d_data2) {

   if( TransformExec(*plan, *d_data, *d_data2, HIPFFT_FORWARD)!= HIPFFT_SUCCESS){
      printf("error in 2D forward transform\n");
   }

}

extern "C" void cuda_2d_inverse_(hipfftHandle *plan,
                Complex **d_data, Complex **d_data2) {

   if( TransformExec(*plan, *d_data, *d_data2, HIPFFT_BACKWARD)!= HIPFFT_SUCCESS){
      printf("error in 2D inverse transform\n");
   }

}

/************ 3D transform *************/

extern "C" void cuda_3d_plan_(int *NX_p, int *NY_p, int *NZ_p,
                 hipfftHandle *plan) {

 int NX = *NX_p;
 int NY = *NY_p;
 int NZ = *NZ_p;

 int n[3] = { NZ, NY, NX };
 if(hipfftPlanMany(plan, 3, n,
              NULL, 1, NX*NY*NZ,
              NULL, 1, NX*NY*NZ, Transform, 1) != HIPFFT_SUCCESS)
      printf("Error creating plan\n");
}

extern "C" void cuda_3d_forward_(hipfftHandle *plan,
                Complex **d_data, Complex **d_data2) {

   if( TransformExec(*plan, *d_data, *d_data2, HIPFFT_FORWARD)!= HIPFFT_SUCCESS){
      printf("error in 3D forward transform\n");
   }

}

extern "C" void cuda_3d_inverse_(int *NX_p, int *NY_p, int *NZ_p ,hipfftHandle *plan,
                Complex **d_data, Complex **d_data2) {

   int NX = *NX_p;
   int NY = *NY_p;
   int NZ = *NZ_p;

   if( TransformExec(*plan, *d_data, *d_data2, HIPFFT_BACKWARD)!= HIPFFT_SUCCESS){
      printf("error in 3D inverse transform\n");
   }

   // scale kernel paramters
   int nThreads = NX;
   dim3 nBlocks(NY,NZ,1);

   scale_kernel <<< nBlocks, nThreads >>> (NX,NY,NZ,*d_data2, 1.0/double(NX*NY*NZ));
}

/************ 3D Poisson Solver for periodic boundary *************/

extern "C" void cuda_3d_psolver_per_plan_(int *NX_p, int *NY_p, int *NZ_p,
                 hipfftHandle *plan, hipfftHandle *plan1) {

 int NX = *NX_p;
 int NY = *NY_p;
 int NZ = *NZ_p;

 int n[3] = { NZ, NY, NX };
 if(hipfftPlanMany(plan, 3, n,
              NULL, 1, NX*NY*NZ,
              NULL, 1, NX*NY*NZ, HIPFFT_D2Z, 1) != HIPFFT_SUCCESS)
      printf("Error creating plan\n");

 if(hipfftPlanMany(plan1, 3, n,
              NULL, 1, NX*NY*NZ,
              NULL, 1, NX*NY*NZ, HIPFFT_Z2D, 1) != HIPFFT_SUCCESS)
      printf("Error creating plan\n");

}


extern "C" void cuda_3d_psolver_per_(int *NX_p, int *NY_p, int *NZ_p,hipfftHandle *plan,
             hipfftHandle *plan1, Complex **d_data, Complex **d_data2, Real **d_kernel, Real *scal_p,
	     int *geo_p) {

 int NX = *NX_p;
 int NY = *NY_p;
 int NZ = *NZ_p;

 int geo = *geo_p;

 Real scal = *scal_p;

 // multiply kernel paramters
 int nThreads = NX/2+1;
 dim3 nBlocks(NY,NZ,1);

 // copy kernel paramters
 int nthreads = NX/2;
 dim3 nblocks(NY/2,NZ/2,1);

 Complex* dst = *d_data;
 Complex* src = *d_data2;

   if (geo==0) {
    src = *d_data;
    dst = *d_data2;
    zero <<< nblocks, nthreads >>> (NX,NY,NZ, (Real*)dst);
    copy_0 <<< nblocks, nthreads  >>> (NX,NY,NZ, (Real*)src, (Real*)dst);
   }

   // Forward FFT

   if( hipfftExecD2Z(*plan, (Real*)dst, src)!= HIPFFT_SUCCESS){
      printf("error in PSper forward transform\n");
   }

   // multiply with kernel

   multiply_kernel <<< nBlocks, nThreads >>> (NX/2+1,NY,NZ,src,*d_kernel,scal);

   // Inverse FFT

   if( hipfftExecZ2D(*plan1, src, (Real*)dst)!= HIPFFT_SUCCESS){
      printf("error in PSper inverse transform\n");
   }

   if (geo==0)
     copy <<< nblocks, nthreads >>> (NX,NY,NZ, (Real*)dst, (Real*)src);
}

/************ 3D Poisson Solver for general boundary *************/

extern "C" void cuda_3d_psolver_general_plan_(int *NX_p, int *NY_p, int *NZ_p,
                 hipfftHandle *plan1, hipfftHandle *plan1_, hipfftHandle *plan2,
                 hipfftHandle *plan3, hipfftHandle *plan3_, int *switch_alg,
		 int *geo1_p, int *geo2_p, int *geo3_p) {

 int NX = *NX_p;
 int NY = *NY_p;
 int NZ = *NZ_p;

 //int geo1 = *geo1_p;
 int geo2 = *geo2_p;
 int geo3 = *geo3_p;

 int n1d[3]= {1, 1, 1};

 int ysize = NY/2 + geo2 * NY/2;
 int zsize = NZ/2 + geo3 * NZ/2;

 n1d[0] = NX;
 if(hipfftPlanMany(plan1,  1, n1d,
              NULL, 1, NX,
              NULL, 1, NX, HIPFFT_D2Z, ysize*zsize) != HIPFFT_SUCCESS)
      printf("Error creating plan\n");

 if(hipfftPlanMany(plan1_,  1, n1d,
              NULL, 1, NX,
              NULL, 1, NX, HIPFFT_Z2D, ysize*zsize) != HIPFFT_SUCCESS)
      printf("Error creating plan\n");

 n1d[0] = NY;
 if(hipfftPlanMany(plan2,  1, n1d,
              NULL, 1, NY,
              NULL, 1, NY, Transform, (NX/2+1)*zsize) != HIPFFT_SUCCESS)
      printf("Error creating plan\n");

 n1d[0] = NZ;
 if(hipfftPlanMany(plan3,  1, n1d,
              NULL, 1, NZ,
              NULL, 1, NZ, Transform, (NX/2+1)*NY) != HIPFFT_SUCCESS)
      printf("Error creating plan\n");

 int nPrimeSize = 17;
 int primeSize[] = {92,104,116,124,136,148,152,164,172,184,188,204,208,220,228,232,248};

 *switch_alg = 0;
 for (int p=0; p<nPrimeSize; p++)
   if (NZ == primeSize[p]) {
     *switch_alg = 1;
     break;
   }

 n1d[0] = NZ;

 int inembed[1];
 int onembed[1];
 inembed[0] = 1;
 onembed[0] = 1;
 if(hipfftPlanMany(plan3_,  1, n1d,
              inembed, NY, 1,
              onembed, NY, 1, Transform, NY) != HIPFFT_SUCCESS)
      printf("Error creating plan\n");

}

extern "C" void cuda_3d_psolver_general_(int *NX_p, int *NY_p, int *NZ_p,
          hipfftHandle *plan1, hipfftHandle *plan1_, hipfftHandle *plan2,
          hipfftHandle *plan3, hipfftHandle *plan3_,
          Complex **d_data, Complex **d_data2, Real **d_kernel, int *switch_alg,
          int *geo1_p, int *geo2_p, int *geo3_p, Real *scal_p) {

 int NX = *NX_p;
 int NY = *NY_p;
 int NZ = *NZ_p;

 Real scal = *scal_p;

 int geo1 = *geo1_p;
 int geo2 = *geo2_p;
 int geo3 = *geo3_p;

 int ysize=NY/2+geo2*NY/2;
 int zsize=NZ/2+geo3*NZ/2;

 // transpose kernel parameters
 dim3 grid((NX+TILE_DIM-1)/TILE_DIM,(ysize*zsize+TILE_DIM-1)/TILE_DIM,1);
 dim3 threads(TILE_DIM,TILE_DIM,1);

 // spread kernel parameters
 dim3 nblocks(zsize,ysize,1);

 // multiply kernel paramters
 int nThreads = NX/2+1;
 dim3 nBlocks(NZ,NY,1);

 Complex* dst = *d_data;
 Complex* src = *d_data2;

 // X transform 

   if (geo1==0) {
     src = *d_data;
     dst = *d_data2;
     spread<<<nblocks, NX>>>((Real*)src, NX/2, (Real*)dst, NX);
   }

   if( hipfftExecD2Z(*plan1, (Real*)dst, src)!= HIPFFT_SUCCESS){
      printf("error in PSolver forward transform 1\n");
   }

   if (geo2==0) {
     transpose_spread <<< grid, threads >>>(src, dst,NX/2+1,ysize*zsize,NY/2);
   } else {
     transpose <<< grid, threads >>>(src, dst,NX/2+1,ysize*zsize);
   }

   // Y transform
   if( TransformExec(*plan2, dst, src, HIPFFT_FORWARD)!= HIPFFT_SUCCESS){
      printf("error in PSolver forward transform 2\n");
   }

  // Z transform, on entire cube
  if (!(*switch_alg)) {
   grid.x = (NY+TILE_DIM-1)/TILE_DIM;
   grid.y = ((NX/2+1)*zsize+TILE_DIM-1)/TILE_DIM;

   if (geo3==0) {
     transpose_spread <<< grid, threads >>>(src, dst,NY,(NX/2+1)*NZ/2,NZ/2);
   } else {
     transpose <<< grid, threads >>>(src, dst,NY,(NX/2+1)*NZ);
   }

   if( TransformExec(*plan3, dst, src, HIPFFT_FORWARD)!= HIPFFT_SUCCESS){
      printf("error in PSolver forward transform 3\n");
   }
  }
  else {
   if (geo3==0) {
      nblocks.x=zsize;
      nblocks.y=NX;
      spread_y<<<nblocks, NY>>>(src, dst);
   }

   for(int k=0; k<NX; ++k){
     if( TransformExec(*plan3_, dst, src, HIPFFT_FORWARD)!= HIPFFT_SUCCESS){
      printf("error in PSolver forward transform 3\n");
     }
     src += NY*NZ;
     dst += NY*NZ;
   }

   src -= NX*NY*NZ;
   dst -= NX*NY*NZ;
  }

  // multiply with kernel

  multiply_kernel <<< nBlocks, nThreads >>> (NX/2+1,NY,NZ,src,*d_kernel,scal);

  // inverse transform

  // Z transform, on entire cube 
  if (!(*switch_alg)) {
   if( TransformExec(*plan3, src, dst, HIPFFT_BACKWARD)!= HIPFFT_SUCCESS){
      printf("error in PSolver inverse transform 1\n");
   }

   grid.x = (zsize*(NX/2+1)+TILE_DIM-1)/TILE_DIM;
   grid.y = (NY+TILE_DIM-1)/TILE_DIM;

   if (geo3==0) {
     transpose_spread_i <<< grid, threads >>>(dst,src,NZ/2*(NX/2+1),NY,NZ/2);
   } else {
     transpose <<< grid, threads >>>(dst, src,NZ*(NX/2+1),NY);
   }

  }
  else {

   for(int k=0; k<NX; ++k){
     if( TransformExec(*plan3_, src, dst, HIPFFT_BACKWARD)!= HIPFFT_SUCCESS){
      printf("error in PSolver inverse transform 3\n");
     }
     src += NY*NZ;
     dst += NY*NZ;
   }

   src -= NX*NY*NZ;
   dst -= NX*NY*NZ;

   if (geo3==0)
      spread_y_i<<<nblocks, NY>>>(dst, src);
  }

  // Y transform

   if( TransformExec(*plan2, src, dst, HIPFFT_BACKWARD)!= HIPFFT_SUCCESS){
      printf("error in PSolver inverse transform 2\n");
   }

   grid.x = (ysize*zsize+TILE_DIM-1)/TILE_DIM;
   grid.y = (NX/2+1+TILE_DIM-1)/TILE_DIM;

   if (geo2==0) {
      transpose_spread_i <<< grid, threads >>>(dst, src,ysize*zsize,NX/2+1, NY/2);
   } else
      transpose <<< grid, threads >>>(dst, src,ysize*zsize,NX/2+1);

   // X transform

   if( hipfftExecZ2D(*plan1_, src, (Real*)dst)!= HIPFFT_SUCCESS){
      printf("error in PSolver inverse transform 3\n");
   }

   nblocks.x=zsize;
   nblocks.y=ysize;
   if (geo1==0) {
      spread_i<<<nblocks, NX/2>>>((Real*)dst, NX/2, (Real*)src, NX);
   }

   //scale_kernel<<< nBlocks, nThreads >>> (NX/2+1,NY,NZ,(Real*)dst,scal); 
}


