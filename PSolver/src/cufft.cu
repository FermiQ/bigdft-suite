#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include "hipfft/hipfft.h"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"
#include "config.h"
 
#define DOUBLE

#ifdef DOUBLE
#define Complex  hipfftDoubleComplex
#define Real double
#define Transform HIPFFT_Z2Z
#define TransformExec hipfftExecZ2Z
#else
#define Complex  hipfftComplex
#define Real float
#define Transform HIPFFT_C2C
#define TransformExec hipfftExecC2C
#endif

#define TILE_DIM  8


#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

// synchronize blocks
extern "C" void synchronize_() {
 
  hipDeviceSynchronize();
}

// allocate device memory
extern "C" void FC_FUNC(cudamalloc, CUDAMALLOC) (int *size, Real **d_data,int *ierr) {

  *ierr = hipMalloc((void**)d_data, sizeof(Real)*(*size));
  if( hipGetLastError() != hipSuccess)
      printf("allocate error\n");
}

// allocate device memory
extern "C" void FC_FUNC(cudamemset, CUDAMEMSET) (Real **d_data, int* value, int* size,int *ierr) {

  *ierr = hipMemset((void*)*d_data, *value, sizeof(Real)*(*size));
  if( hipGetLastError() != hipSuccess)
      printf("cudamemset error\n");
}

extern "C" void FC_FUNC(cudafree, CUDAFREE) (Real **d_data) {

  hipFree(*d_data);
}

extern "C" void FC_FUNC(cufftdestroy, CUFFTDESTROY) (hipfftHandle *plan) {

  hipfftDestroy(*plan);
}

// set device memory
extern "C" void FC_FUNC_(reset_gpu_data, RESET_GPU_DATA)(int *size, Real* h_data, Real **d_data){

 hipMemcpy(*d_data, h_data, sizeof(Real)*(*size),
         hipMemcpyHostToDevice);
 if( hipGetLastError() != hipSuccess)
      printf("transfer error\n");

}

// read device memory
extern "C" void FC_FUNC_(get_gpu_data, GET_GPU_DATA)(int *size, Real *h_data, Real **d_data) {

 hipMemcpy(h_data, *d_data, sizeof(Real)*(*size),
         hipMemcpyDeviceToHost);
 if (hipGetLastError() != hipSuccess)
        printf("transfer back error\n");
}


// transpose
__global__ void transpose(Complex *idata, Complex *odata,
        int width, int height)
{
  __shared__ Complex tile[TILE_DIM][TILE_DIM+1];

  int xIndex = blockIdx.x * TILE_DIM + threadIdx.x;
  int yIndex = blockIdx.y * TILE_DIM + threadIdx.y;
  int index_in = xIndex + (yIndex)*(width);
  int xIndex1 = blockIdx.y * TILE_DIM + threadIdx.x;
  int yIndex1 = blockIdx.x * TILE_DIM + threadIdx.y;
  int index_out = xIndex1 + (yIndex1)*height;

  if (xIndex < width && yIndex < height)
      tile[threadIdx.y][threadIdx.x] = idata[index_in];
    __syncthreads();

  if (xIndex1 < height && yIndex1 < width) {
      odata[index_out] = tile[threadIdx.x][threadIdx.y];
  }
}

// transpose together with spread operation
__global__ void transpose_spread(Complex *idata, Complex *odata, 
	int width, int height, int bign_h)
{
  __shared__ Complex tile[TILE_DIM][TILE_DIM+1];

  int xIndex = blockIdx.x * TILE_DIM + threadIdx.x;
  int yIndex = blockIdx.y * TILE_DIM + threadIdx.y;
  int index_in = xIndex + (yIndex)*(width);
  int xIndex1 = blockIdx.y * TILE_DIM + threadIdx.x;
  int yIndex1 = blockIdx.x * TILE_DIM + threadIdx.y;
  int index_out = xIndex1 + (yIndex1)*height;
  int div = index_out / bign_h;
  int mod = index_out % bign_h;
  index_out = div * (bign_h << 1) + mod+bign_h;
  int plus = -bign_h;

  if (xIndex < width && yIndex < height)
      tile[threadIdx.y][threadIdx.x] = idata[index_in];
    __syncthreads();

  if (xIndex1 < height && yIndex1 < width) {
      odata[index_out] = tile[threadIdx.x][threadIdx.y];
    #ifdef DOUBLE
      odata[index_out + plus] = make_double2(0., 0.);
    #else
      odata[index_out + plus] = make_float2(0.f, 0.f);
    #endif
  }
}

// transpose together with inverse spread operation
__global__ void transpose_spread_i(Complex *idata, Complex *odata,
        int width, int height, int bign_h)
{
  __shared__ Complex tile[TILE_DIM][TILE_DIM+1];

  int xIndex = blockIdx.x * TILE_DIM + threadIdx.x;
  int yIndex = blockIdx.y * TILE_DIM + threadIdx.y;
  int index_in = xIndex + (yIndex)*(width);
  int xIndex1 = blockIdx.y * TILE_DIM + threadIdx.x;
  int yIndex1 = blockIdx.x * TILE_DIM + threadIdx.y;
  int index_out = xIndex1 + (yIndex1)*height;
  int div = index_in / bign_h;
  int mod = index_in % bign_h;
  index_in = div * (bign_h << 1) + mod;

  if (xIndex < width && yIndex < height)
      tile[threadIdx.y][threadIdx.x] = idata[index_in];
    __syncthreads();

  if (xIndex1 < height && yIndex1 < width)
      odata[index_out] = tile[threadIdx.x][threadIdx.y];
}

// spread operation
__global__ void spread(Real* src, unsigned int spitch, Real* dst, unsigned int dpitch)
{
   unsigned int bid = blockIdx.y * gridDim.x + blockIdx.x;
   unsigned int tid = threadIdx.x;
 
   Real res = (tid >= spitch) ? src[bid * spitch + tid-spitch] : 0.0;
   if( tid < dpitch) {
	dst[bid * dpitch + tid] = res;
   }
}

// inverse spread operation
__global__ void spread_i(Real* src, unsigned int spitch, Real* dst, unsigned int dpitch)
{
   unsigned int bid = blockIdx.y * gridDim.x + blockIdx.x;
   unsigned int tid = threadIdx.x;

   Real res = src[bid * dpitch + tid];
   if( tid < dpitch) dst[bid * spitch + tid] = res;
}

// spread operation for 2nd dim
__global__ void spread_y(Complex* src, Complex* dst)
{
   unsigned int tid = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x + threadIdx.x;
   unsigned int tid1 = (blockIdx.y * gridDim.x * 2 + blockIdx.x) * blockDim.x + threadIdx.x;

   Complex res =  src[tid];
   dst[tid1 + blockDim.x*gridDim.x] = res;
#ifdef DOUBLE
   dst[tid1] = make_double2(0., 0.);
#else
   dst[tid1] = make_float2(0.f, 0.f);
#endif
}

__global__ void spread_y_r(Real* src, Real* dst)
{
   unsigned int tid = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x + threadIdx.x;
   unsigned int tid1 = (blockIdx.y * gridDim.x * 2 + blockIdx.x) * blockDim.x + threadIdx.x;

   Real res =  src[tid];
   dst[tid1 + blockDim.x*gridDim.x] = res;
#ifdef DOUBLE
   dst[tid1] = 0.;
#else
   dst[tid1] = 0.f;
#endif
}

__global__ void spread_z(Real* src, Real* dst)
{
   unsigned int tid = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x + threadIdx.x;
   //unsigned int tid1 = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x + threadIdx.x;

   Real res =  src[tid];
   src[tid + (gridDim.y * gridDim.x) * blockDim.x] = res;
#ifdef DOUBLE
   src[tid] = 0.0;
#else
   src[tid] = 0.f;
#endif
}


// inverse spread operation for 2nd dim
__global__ void spread_y_i(Complex* src, Complex* dst)
{
   unsigned int tid = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x + threadIdx.x;
   unsigned int tid1 = (blockIdx.y * gridDim.x * 2 + blockIdx.x) * blockDim.x + threadIdx.x;

   Complex res =  src[tid1];
   dst[tid] = res;
}

__global__ void spread_y_i_r(Real* src, Real* dst)
{
   unsigned int tid = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x + threadIdx.x;
   unsigned int tid1 = (blockIdx.y * gridDim.x * 2 + blockIdx.x) * blockDim.x + threadIdx.x;

   Real res =  src[tid1];
   dst[tid] = res;
}


// multiply with potential
__global__ void multiply_kernel(int nx, int ny, int nz, Complex *d_data, Real *d_kernel, Real scal) {

 int tj = threadIdx.x;
 int td = blockDim.x;

 int blockData = (nx*ny*nz)/(gridDim.x*gridDim.y);

 int jj = (blockIdx.y*gridDim.x + blockIdx.x)*blockData;

 for (int k=0; k<blockData/td; k++) {
     d_data[jj + tj+ k*td].x *= d_kernel[jj + tj+ k*td]*scal;
     d_data[jj + tj+ k*td].y *= d_kernel[jj + tj+ k*td]*scal;
 }

}

// scale
__global__ void scale_kernel(int nx, int ny, int nz, Complex *d_data, Real mult) {

 int tj = threadIdx.x;
 int td = blockDim.x;

 int blockData = (nx*ny*nz)/(gridDim.x*gridDim.y);

 int jj = (blockIdx.y*gridDim.x + blockIdx.x)*blockData;

 for (int k=0; k<blockData/td; k++) {
     d_data[jj + tj+ k*td].x *= mult;
     d_data[jj + tj+ k*td].y *= mult;
 }

}

__global__ void zero(int nx, int ny, int nz, Real *z) {

        int tj = threadIdx.x;
        int td = blockDim.x;

	int blockData = (nx*ny*nz)/(gridDim.x*gridDim.y);

        int jj = ((blockIdx.y)*gridDim.x + (blockIdx.x))*blockData;

        for (int k=0; k<blockData/td; k++) {
        	z[jj + tj+ k*td] = 0.0;
        }
}

__global__ void copy_0(int nx, int ny, int nz, Real *in, Real *out) {

	int tj = threadIdx.x;
        int td = blockDim.x;

        int jj =  (blockIdx.y*nx*ny/4 + blockIdx.x*nx/2);
        int jj1 =  ((blockIdx.y+nz/2)*nx*ny + (blockIdx.x+ny/2)*nx);


        out[jj1+tj+td] = in[jj+tj];

}

__global__ void copy(int nx,int ny,int nz, Real *in, Real *out) {

        int tj = threadIdx.x;
        //int td = blockDim.x;

        int jj =  (blockIdx.y*nx*ny/4 + blockIdx.x*nx/2);
        int jj1 =  ((blockIdx.y)*nx*ny + (blockIdx.x)*nx);

        out[jj+tj] = in[jj1+tj];
}

/************ 1D transform *************/

extern "C" void cuda_1d_plan_(int *NX_p, int *Nbatch_p,
                 hipfftHandle *plan) {

 int NX = *NX_p;
 int Nbatch = *Nbatch_p;

 int n1d[3]= {NX, 1, 1};

 if(hipfftPlanMany(plan,  1, n1d,
              NULL, 1, NX,
              NULL, 1, NX, Transform, Nbatch) != HIPFFT_SUCCESS)
      printf("Error creating plan\n");

 //hipfftPlan1d(plan, NX, Transform, Nbatch );

}

extern "C" void cuda_1d_forward_(hipfftHandle *plan,
                Complex **d_data, Complex **d_data2) {

   if( TransformExec(*plan, *d_data, *d_data2, HIPFFT_FORWARD)!= HIPFFT_SUCCESS){
      printf("error in 1D forward transform\n");
   }

}

extern "C" void cuda_1d_inverse_(hipfftHandle *plan,
                Complex **d_data, Complex **d_data2) {

   if( TransformExec(*plan, *d_data, *d_data2, HIPFFT_BACKWARD)!= HIPFFT_SUCCESS){
      printf("error in 1D inverse transform\n");
   }

}

/************ 2D transform *************/

extern "C" void cuda_2d_plan_(int *NX_p, int *NY_p, int *Nbatch_p,
                 hipfftHandle *plan) {

 int NX = *NX_p;
 int NY = *NY_p;
 int Nbatch = *Nbatch_p;

 int n1d[3]= {NX, NY, 1};

 if(hipfftPlanMany(plan,  1, n1d,
              NULL, 1, NX*NY,
              NULL, 1, NX*NY, Transform, Nbatch) != HIPFFT_SUCCESS)
      printf("Error creating plan\n");

}

extern "C" void cuda_2d_forward_(hipfftHandle *plan,
                Complex **d_data, Complex **d_data2) {

   if( TransformExec(*plan, *d_data, *d_data2, HIPFFT_FORWARD)!= HIPFFT_SUCCESS){
      printf("error in 2D forward transform\n");
   }

}

extern "C" void cuda_2d_inverse_(hipfftHandle *plan,
                Complex **d_data, Complex **d_data2) {

   if( TransformExec(*plan, *d_data, *d_data2, HIPFFT_BACKWARD)!= HIPFFT_SUCCESS){
      printf("error in 2D inverse transform\n");
   }

}

/************ 3D transform *************/

extern "C" void cuda_3d_plan_(int *NX_p, int *NY_p, int *NZ_p,
                 hipfftHandle *plan) {

 int NX = *NX_p;
 int NY = *NY_p;
 int NZ = *NZ_p;

 int n[3] = { NZ, NY, NX };
 if(hipfftPlanMany(plan, 3, n,
              NULL, 1, NX*NY*NZ,
              NULL, 1, NX*NY*NZ, Transform, 1) != HIPFFT_SUCCESS)
      printf("Error creating plan\n");
}

extern "C" void cuda_3d_forward_(hipfftHandle *plan,
                Complex **d_data, Complex **d_data2) {

   if( TransformExec(*plan, *d_data, *d_data2, HIPFFT_FORWARD)!= HIPFFT_SUCCESS){
      printf("error in 3D forward transform\n");
   }

}

extern "C" void cuda_3d_inverse_(int *NX_p, int *NY_p, int *NZ_p ,hipfftHandle *plan,
                Complex **d_data, Complex **d_data2) {

   int NX = *NX_p;
   int NY = *NY_p;
   int NZ = *NZ_p;

   if( TransformExec(*plan, *d_data, *d_data2, HIPFFT_BACKWARD)!= HIPFFT_SUCCESS){
      printf("error in 3D inverse transform\n");
   }

   // scale kernel paramters
   int nThreads = NX;
   dim3 nBlocks(NY,NZ,1);

   scale_kernel <<< nBlocks, nThreads >>> (NX,NY,NZ,*d_data2, 1.0/double(NX*NY*NZ));
}

/************ 3D Poisson Solver for periodic boundary *************/

extern "C" void cuda_3d_psolver_cufft3d_plan_(int *NX_p, int *NY_p, int *NZ_p,
                 hipfftHandle *plan, hipfftHandle *plan1) {

 int NX = *NX_p;
 int NY = *NY_p;
 int NZ = *NZ_p;

 int n[3] = { NZ, NY, NX };
 if(hipfftPlanMany(plan, 3, n,
              NULL, 1, NX*NY*NZ,
              NULL, 1, NX*NY*NZ, HIPFFT_D2Z, 1) != HIPFFT_SUCCESS)
      printf("Error creating plan\n");

 if(hipfftPlanMany(plan1, 3, n,
              NULL, 1, NX*NY*NZ,
              NULL, 1, NX*NY*NZ, HIPFFT_Z2D, 1) != HIPFFT_SUCCESS)
      printf("Error creating plan\n");

}


extern "C" void cuda_3d_psolver_cufft3d_(int *NX_p, int *NY_p, int *NZ_p,hipfftHandle *plan,
             hipfftHandle *plan1, Complex **d_data, Complex **d_data2, Real **d_kernel, Real *scal_p,
	     int *geo1_p, int *geo2_p, int *geo3_p) {

 int NX = *NX_p;
 int NY = *NY_p;
 int NZ = *NZ_p;

 int geo1 = *geo1_p;
 int geo2 = *geo2_p;
 int geo3 = *geo3_p;

 int ysize = NY/2 + geo2 * NY/2;
 int zsize = NZ/2 + geo3 * NZ/2;

 Real scal = *scal_p;

 // multiply kernel paramters
 int nThreads = NX/2+1;
 dim3 nBlocks(NY,NZ,1);

 // copy kernel paramters
 int nthreads = NX/2;
 dim3 nblocks(NY/2,NZ/2,1);

 // spread kernel parameters
 dim3 nblocks_s(zsize,ysize,1);

 Complex* dst = *d_data;
 Complex* src = *d_data2;

   if (geo1==0 && geo2==0 && geo3==0) {
    src = *d_data;
    dst = *d_data2;
    zero <<< nblocks, nthreads >>> (NX,NY,NZ, (Real*)dst);
    copy_0 <<< nblocks, nthreads  >>> (NX,NY,NZ, (Real*)src, (Real*)dst);
   }
   else {
    if (geo1==0) {
     src = *d_data;
     dst = *d_data2;
     spread<<<nblocks_s, NX>>>((Real*)src, NX/2, (Real*)dst, NX);
    }
    if (geo2==0) {
      if (geo1==0) {
        src = *d_data2;
        dst = *d_data;
      } else {
        src = *d_data;
        dst = *d_data2;
      }
      nblocks_s.x=ysize;
      nblocks_s.y=zsize;
      spread_y_r<<<nblocks_s, NX>>>((Real*)src, (Real*)dst);
    }
    if (geo3==0) {
      nblocks_s.x=NY;
      nblocks_s.y=zsize;
      spread_z<<<nblocks_s, NX>>>((Real*)dst, (Real*)src);
    }
   }

   // Forward FFT

   if( hipfftExecD2Z(*plan, (Real*)dst, src)!= HIPFFT_SUCCESS){
      printf("error in PSper forward transform\n");
   }

   // multiply with kernel

   multiply_kernel <<< nBlocks, nThreads >>> (NX/2+1,NY,NZ,src,*d_kernel,scal);

   // Inverse FFT

   if( hipfftExecZ2D(*plan1, src, (Real*)dst)!= HIPFFT_SUCCESS){
      printf("error in PSper inverse transform\n");
   }

   if (geo1==0 && geo2==0 && geo3==0)
     copy <<< nblocks, nthreads >>> (NX,NY,NZ, (Real*)dst, (Real*)src);
   else { 
    if (geo2==0) {
       nblocks_s.x=ysize;
       nblocks_s.y=zsize;
       spread_y_i_r<<<nblocks_s, NX>>>((Real*)dst, (Real*)src);
    }
    if (geo1==0) {
       if (geo2==0) {
        Complex* tmp = src;
        src = dst;
        dst = tmp;
       }
      nblocks_s.x=zsize;
      nblocks_s.y=ysize; 
      spread_i<<<nblocks_s, NX/2>>>((Real*)dst, NX/2, (Real*)src, NX);
    }
   }
}

/************ 3D Poisson Solver for general boundary *************/

extern "C" void FC_FUNC_(cuda_3d_psolver_general_plan, CUDA_3D_PSOLVER_GENERAL_PLAN)(int *N,
                 hipfftHandle *plan, int *switch_alg,
		 int *geo) {

 int NX = N[0];
 int NY = N[1];
 int NZ = N[2];

 //int geo1 = geo[0];
 int geo2 = geo[1];
 int geo3 = geo[2];

 int n1d[3]= {1, 1, 1};

 int ysize = NY/2 + geo2 * NY/2;
 int zsize = NZ/2 + geo3 * NZ/2;

 n1d[0] = NX;
 if(hipfftPlanMany(plan,  1, n1d,
              NULL, 1, NX,
              NULL, 1, NX, HIPFFT_D2Z, ysize*zsize) != HIPFFT_SUCCESS)
      printf("Error creating plan\n");

 if(hipfftPlanMany(plan+1,  1, n1d,
              NULL, 1, NX,
              NULL, 1, NX, HIPFFT_Z2D, ysize*zsize) != HIPFFT_SUCCESS)
      printf("Error creating plan\n");

 n1d[0] = NY;
 if(hipfftPlanMany(plan+2,  1, n1d,
              NULL, 1, NY,
              NULL, 1, NY, Transform, (NX/2+1)*zsize) != HIPFFT_SUCCESS)
      printf("Error creating plan\n");

 n1d[0] = NZ;
 if(hipfftPlanMany(plan+3,  1, n1d,
              NULL, 1, NZ,
              NULL, 1, NZ, Transform, (NX/2+1)*NY) != HIPFFT_SUCCESS)
      printf("Error creating plan\n");

 *switch_alg = 0;

 /*int nPrimeSize = 17;
 int primeSize[] = {92,104,116,124,136,148,152,164,172,184,188,204,208,220,228,232,248};

 for (int p=0; p<nPrimeSize; p++)
   if (NZ == primeSize[p]) {
     *switch_alg = 1;
     break;
   }

 n1d[0] = NZ;

 int inembed[1];
 int onembed[1];
 inembed[0] = 1;
 onembed[0] = 1;
 if(hipfftPlanMany(plan+4,  1, n1d,
              inembed, NY, 1,
              onembed, NY, 1, Transform, NY) != HIPFFT_SUCCESS)
      printf("Error creating plan\n");*/

}

extern "C" void FC_FUNC_(cuda_3d_psolver_general, CUDA_3D_PSOLVER_GENERAL)(int *N,
          hipfftHandle *plan,
          Complex **d_data, Complex **d_data2, Real **d_kernel, int *switch_alg,
          int *geo, Real *scal_p) {

 int NX = N[0];
 int NY = N[1];
 int NZ = N[2];

 Real scal = *scal_p;

 int geo1 = geo[0];
 int geo2 = geo[1];
 int geo3 = geo[2];

 int ysize=NY/2+geo2*NY/2;
 int zsize=NZ/2+geo3*NZ/2;

 // transpose kernel parameters
 dim3 grid((NX/2+1+TILE_DIM-1)/TILE_DIM,(ysize*zsize+TILE_DIM-1)/TILE_DIM,1);
 dim3 threads(TILE_DIM,TILE_DIM,1);

 // spread kernel parameters
 dim3 nblocks(zsize,ysize,1);

 // multiply kernel paramters
 int nThreads = NX/2+1;
 dim3 nBlocks(NZ,NY,1);

 Complex* dst = *d_data;
 Complex* src = *d_data2;

 // X transform 

   if (geo1==0) {
     src = *d_data;
     dst = *d_data2;
     spread<<<nblocks, NX>>>((Real*)src, NX/2, (Real*)dst, NX);
   }

   if( hipfftExecD2Z(plan[0], (Real*)dst, src)!= HIPFFT_SUCCESS){
      printf("error in PSolver forward transform 1\n");
   }

   if (geo2==0) {
     transpose_spread <<< grid, threads >>>(src, dst,NX/2+1,ysize*zsize,NY/2);
   } else {
     transpose <<< grid, threads >>>(src, dst,NX/2+1,ysize*zsize);
   }

   // Y transform
   if( TransformExec(plan[2], dst, src, HIPFFT_FORWARD)!= HIPFFT_SUCCESS){
      printf("error in PSolver forward transform 2\n");
   }

  // Z transform, on entire cube
  if (!(*switch_alg)) {
   grid.x = (NY+TILE_DIM-1)/TILE_DIM;
   grid.y = ((NX/2+1)*zsize+TILE_DIM-1)/TILE_DIM;

   if (geo3==0) {
     transpose_spread <<< grid, threads >>>(src,dst,NY,(NX/2+1)*NZ/2,NZ/2);
   } else {
     transpose <<< grid, threads >>>(src, dst,NY,(NX/2+1)*NZ);
   }

   if( TransformExec(plan[3], dst, src, HIPFFT_FORWARD)!= HIPFFT_SUCCESS){
      printf("error in PSolver forward transform 3\n");
   }
  }
  else {
   if (geo3==0) {
      nblocks.x=zsize;
      nblocks.y=NX;
      spread_y<<<nblocks, NY>>>(src, dst);
   }

   for(int k=0; k<NX; ++k){
     if( TransformExec(plan[4], dst, src, HIPFFT_FORWARD)!= HIPFFT_SUCCESS){
      printf("error in PSolver forward transform 3\n");
     }
     src += NY*NZ;
     dst += NY*NZ;
   }

   src -= NX*NY*NZ;
   dst -= NX*NY*NZ;
  }

  // multiply with kernel

  multiply_kernel <<< nBlocks, nThreads >>> (NX/2+1,NY,NZ,src,*d_kernel,scal);

  // inverse transform

  // Z transform, on entire cube 
  if (!(*switch_alg)) {
   if( TransformExec(plan[3], src, dst, HIPFFT_BACKWARD)!= HIPFFT_SUCCESS){
      printf("error in PSolver inverse transform 1\n");
   }

   grid.x = (zsize*(NX/2+1)+TILE_DIM-1)/TILE_DIM;
   grid.y = (NY+TILE_DIM-1)/TILE_DIM;

   if (geo3==0) {
     transpose_spread_i <<< grid, threads >>>(dst, src,NZ/2*(NX/2+1),NY,NZ/2);
   } else {
     transpose <<< grid, threads >>>(dst, src,NZ*(NX/2+1),NY);
   }

  }
  else {

   for(int k=0; k<NX; ++k){
     if( TransformExec(plan[4], src, dst, HIPFFT_BACKWARD)!= HIPFFT_SUCCESS){
      printf("error in PSolver inverse transform 3\n");
     }
     src += NY*NZ;
     dst += NY*NZ;
   }

   src -= NX*NY*NZ;
   dst -= NX*NY*NZ;

   if (geo3==0)
      spread_y_i<<<nblocks, NY>>>(dst, src);
  }

  // Y transform

   if( TransformExec(plan[2], src, dst, HIPFFT_BACKWARD)!= HIPFFT_SUCCESS){
      printf("error in PSolver inverse transform 2\n");
   }

   grid.x = (ysize*zsize+TILE_DIM-1)/TILE_DIM;
   grid.y = (NX/2+1+TILE_DIM-1)/TILE_DIM;

   if (geo2==0) {
      transpose_spread_i <<< grid, threads >>>(dst, src,ysize*zsize,NX/2+1, NY/2);
   } else
      transpose <<< grid, threads >>>(dst, src,ysize*zsize,NX/2+1);

   // X transform

   if( hipfftExecZ2D(plan[1], src, (Real*)dst)!= HIPFFT_SUCCESS){
      printf("error in PSolver inverse transform 3\n");
   }

   nblocks.x=zsize;
   nblocks.y=ysize;
   if (geo1==0) {
      spread_i<<<nblocks, NX/2>>>((Real*)dst,NX/2, (Real*)src, NX);
   }
}


extern "C" void FC_FUNC_(cuda_3d_psolver_plangeneral, CUDA_3D_PSOLVER_PLANGENERAL)(int *N,
          Complex **d_data, Complex **d_data2, Real **d_kernel,
          int *geo, Real *scal_p) {

 hipfftHandle plan;

 int NX = N[0];
 int NY = N[1];
 int NZ = N[2];

 Real scal = *scal_p;

 int geo1 = geo[0];
 int geo2 = geo[1];
 int geo3 = geo[2];

 int ysize=NY/2+geo2*NY/2;
 int zsize=NZ/2+geo3*NZ/2;

 // transpose kernel parameters
 dim3 grid((NX/2+1+TILE_DIM-1)/TILE_DIM,(ysize*zsize+TILE_DIM-1)/TILE_DIM,1);
 dim3 threads(TILE_DIM,TILE_DIM,1);

 // spread kernel parameters
 dim3 nblocks(zsize,ysize,1);

 // multiply kernel paramters
 int nThreads = NX/2+1;
 dim3 nBlocks(NZ,NY,1);

 Complex* dst = *d_data;
 Complex* src = *d_data2;

 int n1d[3]= {1, 1, 1};

 n1d[0] = NX;
 if(hipfftPlanMany(&plan,  1, n1d,
              NULL, 1, NX,
              NULL, 1, NX, HIPFFT_D2Z, ysize*zsize) != HIPFFT_SUCCESS)
      printf("Error creating plan 1\n");

 // X transform 

   if (geo1==0) {
     src = *d_data;
     dst = *d_data2;
     spread<<<nblocks, NX>>>((Real*)src, NX/2, (Real*)dst, NX);
   }

   if( hipfftExecD2Z(plan, (Real*)dst, src)!= HIPFFT_SUCCESS){
      printf("error in PSolver forward transform 1\n");
   }

   if (geo2==0) {
     transpose_spread <<< grid, threads >>>(src, dst,NX/2+1,ysize*zsize,NY/2);
   } else {
     transpose <<< grid, threads >>>(src, dst,NX/2+1,ysize*zsize);
   }

   hipfftDestroy(plan);

   n1d[0] = NY;
   if(hipfftPlanMany(&plan,  1, n1d,
              NULL, 1, NY,
              NULL, 1, NY, Transform, (NX/2+1)*zsize) != HIPFFT_SUCCESS)
      printf("Error creating plan 2\n");

   // Y transform
   if( TransformExec(plan, dst, src, HIPFFT_FORWARD)!= HIPFFT_SUCCESS){
      printf("error in PSolver forward transform 2\n");
   }

  // Z transform, on entire cube
   grid.x = (NY+TILE_DIM-1)/TILE_DIM;
   grid.y = ((NX/2+1)*zsize+TILE_DIM-1)/TILE_DIM;

   if (geo3==0) {
     transpose_spread <<< grid, threads >>>(src, dst,NY,(NX/2+1)*NZ/2,NZ/2);
   } else {
     transpose <<< grid, threads >>>(src, dst,NY,(NX/2+1)*NZ);
   }

   hipfftDestroy(plan);
   n1d[0] = NZ;
   if(hipfftPlanMany(&plan,  1, n1d,
              NULL, 1, NZ,
              NULL, 1, NZ, Transform, (NX/2+1)*NY) != HIPFFT_SUCCESS)
      printf("Error creating plan 3\n");

   if( TransformExec(plan, dst, src, HIPFFT_FORWARD)!= HIPFFT_SUCCESS){
      printf("error in PSolver forward transform 3\n");
   }

  // multiply with kernel

  multiply_kernel <<< nBlocks, nThreads >>> (NX/2+1,NY,NZ,src,*d_kernel,scal);

  // inverse transform

  // Z transform, on entire cube 
   if( TransformExec(plan, src, dst, HIPFFT_BACKWARD)!= HIPFFT_SUCCESS){
      printf("error in PSolver inverse transform 1\n");
   }

   grid.x = (zsize*(NX/2+1)+TILE_DIM-1)/TILE_DIM;
   grid.y = (NY+TILE_DIM-1)/TILE_DIM;

   if (geo3==0) {
     transpose_spread_i <<< grid, threads >>>(dst, src,NZ/2*(NX/2+1),NY,NZ/2);
   } else {
     transpose <<< grid, threads >>>(dst, src,NZ*(NX/2+1),NY);
   }

  // Y transform

   hipfftDestroy(plan);
   n1d[0] = NY;
   if(hipfftPlanMany(&plan,  1, n1d,
              NULL, 1, NY,
              NULL, 1, NY, Transform, (NX/2+1)*zsize) != HIPFFT_SUCCESS)
      printf("Error creating plan 4\n");

   if( TransformExec(plan, src, dst,HIPFFT_BACKWARD)!= HIPFFT_SUCCESS){
      printf("error in PSolver inverse transform 2\n");
   }

   grid.x = (ysize*zsize+TILE_DIM-1)/TILE_DIM;
   grid.y = (NX/2+1+TILE_DIM-1)/TILE_DIM;

   if (geo2==0) {
      transpose_spread_i <<< grid, threads >>>(dst,src,ysize*zsize,NX/2+1, NY/2);
   } else
      transpose <<< grid, threads >>>(dst, src,ysize*zsize,NX/2+1);

   // X transform

   hipfftDestroy(plan);
   n1d[0] = NX;
   if(hipfftPlanMany(&plan,  1, n1d,
              NULL, 1, NX,
              NULL, 1, NX, HIPFFT_Z2D, ysize*zsize) != HIPFFT_SUCCESS)
      printf("Error creating plan 5\n");

   if( hipfftExecZ2D(plan, src, (Real*)dst)!= HIPFFT_SUCCESS){
      printf("error in PSolver inverse transform 3\n");
   }

   nblocks.x=zsize;
   nblocks.y=ysize;
   if (geo1==0) {
      spread_i<<<nblocks, NX/2>>>((Real*)dst,NX/2, (Real*)src, NX);
   }

   hipfftDestroy(plan);
}


//Specialization of the computation part for each reduction kernel.
//the kern1_red itself is useless as it is the same for all 3 reductions
//keeping it, as we may want to use another someday

typedef void(*comp_and_red_op)(int, Real*, Real*, Real*, Real*, Real*, Real*, Real*, Real*, Real*, Real*, Real*, Real*, Real*);
typedef void(*red_op)(int, Real*, Real*, Real*, Real*, Real*, Real*, Real*, Real*, Real*, Real*, Real*, Real*, Real*);

__device__
void kern1_comp_and_red (int i , Real* p_GPU, Real* q_GPU, Real* r_GPU, Real* x_GPU, Real* z_GPU, Real* corr_GPU, Real* oneoeps_GPU, Real* alpha_GPU, Real* beta_GPU, Real* beta0_GPU, Real* kappa_GPU, Real* g_odata, Real* sum){
  Real zeta=z_GPU[i]*oneoeps_GPU[i];
  z_GPU[i]=zeta;
  *sum+= (r_GPU[i]*zeta);
}

__device__
void kern1_red (int i , Real* p_GPU, Real* q_GPU, Real* r_GPU, Real* x_GPU, Real* z_GPU, Real* corr_GPU, Real* oneoeps_GPU, Real* alpha_GPU, Real* beta_GPU, Real* beta0_GPU, Real* kappa_GPU, Real* g_odata, Real* sum){
  *sum+= (g_odata[i]);
}


__device__
void kern2_comp_and_red (int i , Real* p_GPU, Real* q_GPU, Real* r_GPU, Real* x_GPU, Real* z_GPU, Real* corr_GPU, Real* oneoeps_GPU, Real* alpha_GPU, Real* beta_GPU, Real* beta0_GPU, Real* kappa_GPU, Real* g_odata, Real* sum){
  Real zeta=z_GPU[i];
  Real pval = zeta+(*beta_GPU / *beta0_GPU)*p_GPU[i];
  Real qval = zeta*corr_GPU[i]+r_GPU[i]+(*beta_GPU / *beta0_GPU)*q_GPU[i];
  p_GPU[i] = pval;
  q_GPU[i] = qval;
  *sum+= (pval*qval);
}

__device__
void kern3_comp_and_red (int i , Real* p_GPU, Real* q_GPU, Real* r_GPU, Real* x_GPU, Real* z_GPU, Real* corr_GPU, Real* oneoeps_GPU, Real* alpha_GPU, Real* beta_GPU, Real* beta0_GPU, Real* kappa_GPU, Real* g_odata, Real* sum){
  x_GPU[i] = x_GPU[i] + *alpha_GPU*p_GPU[i];
  r_GPU[i] = r_GPU[i] - *alpha_GPU*q_GPU[i];
  z_GPU[i] = r_GPU[i] * oneoeps_GPU[i];
  *sum+=(r_GPU[i]*r_GPU[i]);
}




//helper functions for the reduction (reduction taken from NVIDIA cuda samples)
template<class T>
struct SharedMemory
{
    __device__ inline operator       T *()
    {
        extern __shared__ int __smem[];
        return (T *)__smem;
    }

    __device__ inline operator const T *() const
    {
        extern __shared__ int __smem[];
        return (T *)__smem;
    }
};
// specialize for double to avoid unaligned memory
// access compile errors
template<>
struct SharedMemory<double>
{
    __device__ inline operator       double *()
    {
        extern __shared__ double __smem_d[];
        return (double *)__smem_d;
    }

    __device__ inline operator const double *() const
    {
        extern __shared__ double __smem_d[];
        return (double *)__smem_d;
    }
};



/*actual kernel call for the reduction, that is specialized with 2 template 
subkernels, one for computation, the other for specific reduction part.
Result is written in g_odata array in GPU memory. So this must be called several 
times to actually reduce to a single element.
*/
template <unsigned int blockSize, bool nIsPow2, comp_and_red_op op1, red_op op2>
__global__ void
reduce_kernel(int n, int reduceOnly, Real* p_GPU, Real* q_GPU, Real* r_GPU, Real* x_GPU, Real* z_GPU, Real* corr_GPU, Real* oneoeps_GPU, Real* alpha_GPU, Real* beta_GPU, Real* beta0_GPU, Real* kappa_GPU, Real* g_odata)
{
    Real *sdata = SharedMemory<Real>();

    // perform first level of reduction,
    // reading from global memory, writing to shared memory
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x*blockSize*2 + threadIdx.x;
    unsigned int gridSize = blockSize*2*gridDim.x;

    Real mySum = 0;
    // we reduce multiple elements per thread.  The number is determined by the
    // number of active thread blocks (via gridDim).  More blocks will result
    // in a larger gridSize and therefore fewer elements per thread
    while (i < n)
    {
    if(!reduceOnly){
        op1 (i , p_GPU, q_GPU, r_GPU, x_GPU, z_GPU, corr_GPU, oneoeps_GPU, alpha_GPU, beta_GPU, beta0_GPU, kappa_GPU, g_odata, &mySum);
        // ensure we don't read out of bounds -- this is optimized away for powerOf2 sized arrays
        if (nIsPow2 || i + blockSize < n){
            op1 (i + blockSize, p_GPU, q_GPU, r_GPU, x_GPU, z_GPU, corr_GPU, oneoeps_GPU, alpha_GPU, beta_GPU, beta0_GPU, kappa_GPU, g_odata, &mySum);
        }
    }else{

        //subsequent calls to the kernel after the first one don't have to perform 
        // the computations
        op2 (i, p_GPU, q_GPU, r_GPU, x_GPU, z_GPU, corr_GPU, oneoeps_GPU, alpha_GPU, beta_GPU, beta0_GPU, kappa_GPU, g_odata, &mySum);

        // ensure we don't read out of bounds -- this is optimized away for powerOf2 sized arrays
        if (nIsPow2 || i + blockSize < n)
            op2 (i + blockSize, p_GPU, q_GPU, r_GPU, x_GPU, z_GPU, corr_GPU, oneoeps_GPU, alpha_GPU, beta_GPU, beta0_GPU, kappa_GPU, g_odata, &mySum);
    }

        i += gridSize;
    }

    // each thread puts its local sum into shared memory
    sdata[tid] = mySum;
    __syncthreads();


    // do reduction in shared mem
    if ((blockSize >= 512) && (tid < 256))
    {
        sdata[tid] = mySum = mySum + sdata[tid + 256];
    }

    __syncthreads();

    if ((blockSize >= 256) &&(tid < 128))
    {
            sdata[tid] = mySum = mySum + sdata[tid + 128];
    }

     __syncthreads();

    if ((blockSize >= 128) && (tid <  64))
    {
       sdata[tid] = mySum = mySum + sdata[tid +  64];
    }

    __syncthreads();

#if (__CUDA_ARCH__ >= 300 )
    if ( tid < 32 )
    {
        // Fetch final intermediate sum from 2nd warp
        if (blockSize >=  64) mySum += sdata[tid + 32];
        // Reduce final warp using shuffle
        for (int offset = warpSize/2; offset > 0; offset /= 2) 
        {
            mySum += __shfl_down(mySum, offset);
        }
    }
#else
    // fully unroll reduction within a single warp
    if ((blockSize >=  64) && (tid < 32))
    {
        sdata[tid] = mySum = mySum + sdata[tid + 32];
    }

    __syncthreads();

    if ((blockSize >=  32) && (tid < 16))
    {
        sdata[tid] = mySum = mySum + sdata[tid + 16];
    }

    __syncthreads();

    if ((blockSize >=  16) && (tid <  8))
    {
        sdata[tid] = mySum = mySum + sdata[tid +  8];
    }

    __syncthreads();

    if ((blockSize >=   8) && (tid <  4))
    {
        sdata[tid] = mySum = mySum + sdata[tid +  4];
    }

    __syncthreads();

    if ((blockSize >=   4) && (tid <  2))
    {
        sdata[tid] = mySum = mySum + sdata[tid +  2];
    }

    __syncthreads();

    if ((blockSize >=   2) && ( tid <  1))
    {
        sdata[tid] = mySum = mySum + sdata[tid +  1];
    }

    __syncthreads();
#endif

    // write result for this block to global mem
    if (tid == 0) g_odata[blockIdx.x] = mySum;
}


//wrapper for templated kernel
template <comp_and_red_op op1, red_op op2>
void reduce_step(int s, int threads, int blocks, int reduceOnly,  Real* p_GPU, Real* q_GPU, Real* r_GPU, Real* x_GPU, Real* z_GPU, Real* corr_GPU, Real* oneoeps_GPU, Real* alpha_GPU, Real* beta_GPU, Real* beta0_GPU, Real* kappa_GPU, Real* d_odata){
    //TODO : 2D
    dim3 dimBlock(threads, 1, 1);
    dim3 dimGrid(blocks, 1, 1);
    int smemSize = (threads <= 32) ? 2 * threads * sizeof(Real) : threads * sizeof(Real);

    if (((s&(s-1))==0))//pow2
    {
        switch (threads)
        {
            case 512:
                reduce_kernel<512, true, op1, op2><<< dimGrid, dimBlock, smemSize >>>(s, reduceOnly,  p_GPU, q_GPU, r_GPU, x_GPU, z_GPU, corr_GPU, oneoeps_GPU, alpha_GPU, beta_GPU, beta0_GPU, kappa_GPU, d_odata);
                break;
            case 256:
                reduce_kernel<256, true, op1, op2><<< dimGrid, dimBlock, smemSize >>>(s, reduceOnly,  p_GPU, q_GPU, r_GPU, x_GPU, z_GPU, corr_GPU, oneoeps_GPU, alpha_GPU, beta_GPU, beta0_GPU, kappa_GPU, d_odata);
                break;

            case 128:
                reduce_kernel<128, true, op1, op2><<< dimGrid, dimBlock, smemSize >>>(s, reduceOnly,  p_GPU, q_GPU, r_GPU, x_GPU, z_GPU, corr_GPU, oneoeps_GPU, alpha_GPU, beta_GPU, beta0_GPU, kappa_GPU, d_odata);
                break;
            case 64:
                reduce_kernel<64, true, op1, op2><<< dimGrid, dimBlock, smemSize >>>(s, reduceOnly,  p_GPU, q_GPU, r_GPU, x_GPU, z_GPU, corr_GPU, oneoeps_GPU, alpha_GPU, beta_GPU, beta0_GPU, kappa_GPU, d_odata);
                break;
            case 32:
                reduce_kernel<32, true, op1, op2><<< dimGrid, dimBlock, smemSize >>>(s, reduceOnly,  p_GPU, q_GPU, r_GPU, x_GPU, z_GPU, corr_GPU, oneoeps_GPU, alpha_GPU, beta_GPU, beta0_GPU, kappa_GPU, d_odata);
                break;
            case 16:
                reduce_kernel<16, true, op1, op2><<< dimGrid, dimBlock, smemSize >>>(s, reduceOnly,  p_GPU, q_GPU, r_GPU, x_GPU, z_GPU, corr_GPU, oneoeps_GPU, alpha_GPU, beta_GPU, beta0_GPU, kappa_GPU, d_odata);
                break;
            case  8:
                reduce_kernel<8, true, op1, op2><<< dimGrid, dimBlock, smemSize >>>(s, reduceOnly,  p_GPU, q_GPU, r_GPU, x_GPU, z_GPU, corr_GPU, oneoeps_GPU, alpha_GPU, beta_GPU, beta0_GPU, kappa_GPU, d_odata);
                break;
            case  4:
                reduce_kernel<4, true, op1, op2><<< dimGrid, dimBlock, smemSize >>>(s, reduceOnly,  p_GPU, q_GPU, r_GPU, x_GPU, z_GPU, corr_GPU, oneoeps_GPU, alpha_GPU, beta_GPU, beta0_GPU, kappa_GPU, d_odata);
                break;
            case  2:
                reduce_kernel<2, true, op1, op2><<< dimGrid, dimBlock, smemSize >>>(s, reduceOnly,  p_GPU, q_GPU, r_GPU, x_GPU, z_GPU, corr_GPU, oneoeps_GPU, alpha_GPU, beta_GPU, beta0_GPU, kappa_GPU, d_odata);
                break;
            case  1:
                reduce_kernel<1, true, op1, op2><<< dimGrid, dimBlock, smemSize >>>(s, reduceOnly,  p_GPU, q_GPU, r_GPU, x_GPU, z_GPU, corr_GPU, oneoeps_GPU, alpha_GPU, beta_GPU, beta0_GPU, kappa_GPU, d_odata);
                break;
        }
    }
    else
    {
        switch (threads)
        {
            case 512:
                reduce_kernel<512, false, op1, op2><<< dimGrid, dimBlock, smemSize >>>(s, reduceOnly,  p_GPU, q_GPU, r_GPU, x_GPU, z_GPU, corr_GPU, oneoeps_GPU, alpha_GPU, beta_GPU, beta0_GPU, kappa_GPU, d_odata);
                break;
            case 256:
                reduce_kernel<256, false, op1, op2><<< dimGrid, dimBlock, smemSize >>>(s, reduceOnly,  p_GPU, q_GPU, r_GPU, x_GPU, z_GPU, corr_GPU, oneoeps_GPU, alpha_GPU, beta_GPU, beta0_GPU, kappa_GPU, d_odata);
                break;
            case 128:
                reduce_kernel<128, false, op1, op2><<< dimGrid, dimBlock, smemSize >>>(s, reduceOnly,  p_GPU, q_GPU, r_GPU, x_GPU, z_GPU, corr_GPU, oneoeps_GPU, alpha_GPU, beta_GPU, beta0_GPU, kappa_GPU, d_odata);
                break;
            case 64:
                reduce_kernel<64, false, op1, op2><<< dimGrid, dimBlock, smemSize >>>(s, reduceOnly,  p_GPU, q_GPU, r_GPU, x_GPU, z_GPU, corr_GPU, oneoeps_GPU, alpha_GPU, beta_GPU, beta0_GPU, kappa_GPU, d_odata);
                break;
            case 32:
                reduce_kernel<32, false, op1, op2><<< dimGrid, dimBlock, smemSize >>>(s, reduceOnly,  p_GPU, q_GPU, r_GPU, x_GPU, z_GPU, corr_GPU, oneoeps_GPU, alpha_GPU, beta_GPU, beta0_GPU, kappa_GPU, d_odata);
                break;
            case 16:
                reduce_kernel<16, false, op1, op2><<< dimGrid, dimBlock, smemSize >>>(s, reduceOnly,  p_GPU, q_GPU, r_GPU, x_GPU, z_GPU, corr_GPU, oneoeps_GPU, alpha_GPU, beta_GPU, beta0_GPU, kappa_GPU, d_odata);
                break;
            case  8:
                reduce_kernel<8, false, op1, op2><<< dimGrid, dimBlock, smemSize >>>(s, reduceOnly,  p_GPU, q_GPU, r_GPU, x_GPU, z_GPU, corr_GPU, oneoeps_GPU, alpha_GPU, beta_GPU, beta0_GPU, kappa_GPU, d_odata);
                break;
            case  4:
                reduce_kernel<4, false, op1, op2><<< dimGrid, dimBlock, smemSize >>>(s, reduceOnly,  p_GPU, q_GPU, r_GPU, x_GPU, z_GPU, corr_GPU, oneoeps_GPU, alpha_GPU, beta_GPU, beta0_GPU, kappa_GPU, d_odata);
                break;
            case  2:
                reduce_kernel<2, false, op1, op2><<< dimGrid, dimBlock, smemSize >>>(s, reduceOnly,  p_GPU, q_GPU, r_GPU, x_GPU, z_GPU, corr_GPU, oneoeps_GPU, alpha_GPU, beta_GPU, beta0_GPU, kappa_GPU, d_odata);
                break;
            case  1:
                reduce_kernel<1, false, op1, op2><<< dimGrid, dimBlock, smemSize >>>(s, reduceOnly,  p_GPU, q_GPU, r_GPU, x_GPU, z_GPU, corr_GPU, oneoeps_GPU, alpha_GPU, beta_GPU, beta0_GPU, kappa_GPU, d_odata);
                break;
        }
    }

gpuErrchk( hipPeekAtLastError() );
gpuErrchk( hipDeviceSynchronize() );

//  hipError_t error = hipGetLastError();
//  if(error != hipSuccess)
//  {
//    // print the CUDA error message and exit
//    printf("CUDA error in kernel : %s", hipGetErrorString(error));
//exit(0);
//  }
}

unsigned int nextPow2(unsigned int x)
{
    --x;
    x |= x >> 1;
    x |= x >> 2;
    x |= x >> 4;
    x |= x >> 8;
    x |= x >> 16;
    return ++x;
}

/*this performs some calculations to chose the size of the blocks we want to use 
for reduction, while limiting their number for efficiency purposes, as each kernel
 will handle several elements in this version (see last version of reduction in
reduction sample code from Nvidia)
*/
template <comp_and_red_op op1, red_op op2>
void apply_reduction(int n,
          Real* p_GPU, Real* q_GPU, Real* r_GPU, Real* x_GPU, Real* z_GPU, Real* corr_GPU, Real* oneoeps_GPU, Real* alpha_GPU, Real* beta_GPU, Real* beta0_GPU, Real* kappa_GPU, Real* result) {
    int maxThreads=256;
    int maxBlocks=64;
    int blocks=0;
    int threads=0;

    //get device capability, to avoid block/grid size excceed the upbound
    hipDeviceProp_t prop;
    int device;
    hipGetDevice(&device);
    hipGetDeviceProperties(&prop, device);

    threads = (n < maxThreads*2) ? nextPow2((n + 1)/ 2) : maxThreads;
    blocks = (n + (threads * 2 - 1)) / (threads * 2);

    if ((Real)threads*blocks > (Real)prop.maxGridSize[0] * prop.maxThreadsPerBlock)
    {
        printf("n is too large, please choose a smaller number!\n");
    }

    if (blocks > prop.maxGridSize[0])
    {
        printf("Grid size <%d> excceeds the device capability <%d>, set block size as %d (original %d)\n",
               blocks, prop.maxGridSize[0], threads*2, threads);

        blocks /= 2;
        threads *= 2;
    }

    //we will only use maxblocks blocks, and make each thread work on more data
    blocks = min(maxBlocks, blocks);


    Real *d_odata = NULL;
    hipMalloc((void **) &d_odata, blocks*sizeof(Real));
  if( hipGetLastError() != hipSuccess)
      printf("allocate error\n");
    //first reduction
    hipDeviceSynchronize();
    reduce_step<op1, op2>(n, threads, blocks, 0,  p_GPU, q_GPU, r_GPU, x_GPU, z_GPU, corr_GPU, oneoeps_GPU, alpha_GPU, beta_GPU, beta0_GPU, kappa_GPU, d_odata);
  
  if( hipGetLastError() != hipSuccess)
    printf("result during first reduction kernel\n");

    int s=blocks;
    //loop and perform as many reductions steps as necessary
    while (s > 1)
    {
        threads = (s < maxThreads*2) ? nextPow2((s + 1)/ 2) : maxThreads;
        blocks = (s + (threads * 2 - 1)) / (threads * 2);
        if (blocks > prop.maxGridSize[0])
        {
            printf("Grid size <%d> excceeds the device capability <%d>, set block size as %d (original %d)\n",
            blocks, prop.maxGridSize[0], threads*2, threads);

            blocks /= 2;
            threads *= 2;
        }
        blocks = min(maxBlocks, blocks);

        reduce_step<op1, op2>(s, threads, blocks, 1, p_GPU, q_GPU, r_GPU, x_GPU, z_GPU, corr_GPU, oneoeps_GPU, alpha_GPU, beta_GPU, beta0_GPU, kappa_GPU, d_odata);
        if( hipGetLastError() != hipSuccess)
            printf("result during reduction kernel\n");
        s = (s + (threads*2-1)) / (threads*2);
    }

  //TODO: move result copy to user code ?
  hipMemcpy(result, d_odata, sizeof(Real), hipMemcpyDeviceToHost);
  if( hipGetLastError() != hipSuccess)
    printf("result retrieve error\n");
  hipFree(d_odata);
  hipDeviceSynchronize();
}

//these will be called from fortran, and apply the reduction with the right subkernels

extern "C" void FC_FUNC_(first_reduction_kernel, FIRST_REDUCTION_KERNEL)(int* n1, int* n23,
          Real** p_GPU, Real** q_GPU, Real** r_GPU, Real** x_GPU, Real** z_GPU, Real** corr_GPU, Real** oneoeps_GPU, Real** alpha_GPU, Real** beta_GPU, Real** beta0_GPU, Real** kappa_GPU, Real* result) {

    int n=(*n1) * (*n23);
    apply_reduction<kern1_comp_and_red, kern1_red>(n, *p_GPU, *q_GPU, *r_GPU, *x_GPU, *z_GPU, *corr_GPU, *oneoeps_GPU, *alpha_GPU, *beta_GPU, *beta0_GPU, *kappa_GPU, result);

}

extern "C" void FC_FUNC_(second_reduction_kernel, SECOND_REDUCTION_KERNEL)(int* n1, int* n23,
          Real** p_GPU, Real** q_GPU, Real** r_GPU, Real** x_GPU, Real** z_GPU, Real** corr_GPU, Real** oneoeps_GPU, Real** alpha_GPU, Real** beta_GPU, Real** beta0_GPU, Real** kappa_GPU, Real* result) {

    int n=(*n1) * (*n23);
    apply_reduction<kern2_comp_and_red, kern1_red>(n, *p_GPU, *q_GPU, *r_GPU, *x_GPU, *z_GPU, *corr_GPU, *oneoeps_GPU, *alpha_GPU, *beta_GPU, *beta0_GPU, *kappa_GPU, result);

}

extern "C" void FC_FUNC_(third_reduction_kernel, THIRD_REDUCTION_KERNEL)(int* n1, int* n23,
          Real** p_GPU, Real** q_GPU, Real** r_GPU, Real** x_GPU, Real** z_GPU, Real** corr_GPU, Real** oneoeps_GPU, Real** alpha_GPU, Real** beta_GPU, Real** beta0_GPU, Real** kappa_GPU, Real* result) {

    int n=(*n1) * (*n23);
    apply_reduction<kern3_comp_and_red, kern1_red>(n, *p_GPU, *q_GPU, *r_GPU, *x_GPU, *z_GPU, *corr_GPU, *oneoeps_GPU, *alpha_GPU, *beta_GPU, *beta0_GPU, *kappa_GPU, result);

}
